#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "CellListGPU.cuh"

/*! \file CellListGPU.cu
    \brief Defines GPU kernel code for cell list generation on the GPU
*/

//! Kernel that computes the cell list on the GPU
/*! \param d_cell_size Number of particles in each cell
    \param d_xyzf Cell XYZF data array
    \param d_tdb Cell TDB data array
    \param d_cell_orientation Particle orientation in cell list
    \param d_cell_idx Particle index in cell list
    \param d_conditions Conditions flags for detecting overflow and other error conditions
    \param d_pos Particle position array
    \param d_orientation Particle orientation array
    \param d_charge Particle charge array
    \param d_diameter Particle diameter array
    \param d_body Particle body array
    \param N Number of particles
    \param n_ghost Number of ghost particles
    \param Nmax Maximum number of particles that can be placed in a single cell
    \param flag_charge Set to true to store chage in the flag position in \a d_xyzf
    \param flag_type Set to true to store type in the flag position in \a d_xyzf
    \param box Box dimensions
    \param ci Indexer to compute cell id from cell grid coords
    \param cli Indexer to index into \a d_xyzf and \a d_tdb
    \param ghost_width Width of ghost layer
    
    \note Optimized for Fermi
*/
__global__ void gpu_compute_cell_list_kernel(unsigned int *d_cell_size,
                                             float4 *d_xyzf,
                                             float4 *d_tdb,
                                             float4 *d_cell_orientation,
                                             unsigned int *d_cell_idx,
                                             uint3 *d_conditions,
                                             const float4 *d_pos,
                                             const float4 *d_orientation,
                                             const float *d_charge,
                                             const float *d_diameter,
                                             const unsigned int *d_body,
                                             const unsigned int N,
                                             const unsigned int n_ghost,
                                             const unsigned int Nmax,
                                             const bool flag_charge,
                                             const bool flag_type,
                                             const BoxDim box,
                                             const Index3D ci,
                                             const Index2D cli,
                                             const Scalar3 ghost_width) 
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N + n_ghost)
        return;

    float4 postype = d_pos[idx];
    float3 pos = make_float3(postype.x, postype.y, postype.z);

    float flag = 0.0f;
    float diameter = 0.0f;
    float body = 0;
    float type = postype.w;
    float4 orientation = make_float4(0,0,0,0);
    if (d_tdb != NULL)
        {
        diameter = d_diameter[idx];
        body = __int_as_float(d_body[idx]);
        }
    if (d_cell_orientation != NULL)
        {
        orientation = d_orientation[idx];
        }

    if (flag_charge)
        flag = d_charge[idx];
    else if (flag_type)
        flag = type;
    else
        flag = __int_as_float(idx);

    // check for nan pos
    if (isnan(pos.x) || isnan(pos.y) || isnan(pos.z))
        {
        (*d_conditions).y = idx+1;
        return;
        }

    uchar3 periodic = box.getPeriodic();
    Scalar3 f = box.makeFraction(pos,ghost_width);

    // check if the particle is inside the unit cell + ghost layer
    if ((!periodic.x && (f.x < Scalar(0.0) || f.x >= Scalar(1.0))) ||
        (!periodic.y && (f.y < Scalar(0.0) || f.y >= Scalar(1.0))) ||
        (!periodic.z && (f.z < Scalar(0.0) || f.z >= Scalar(1.0))) )
        {
        // if a ghost particle is out of bounds, silently ignore it
        if (idx < N)
            (*d_conditions).z = idx+1;
        return;
        }

    // find the bin each particle belongs in
    int ib = (int)(f.x * ci.getW());
    int jb = (int)(f.y * ci.getH());
    int kb = (int)(f.z * ci.getD());

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    unsigned int bin = ci(ib, jb, kb);


    unsigned int size = atomicInc(&d_cell_size[bin], 0xffffffff);
    if (size < Nmax)
        {
        unsigned int write_pos = cli(size, bin);
        d_xyzf[write_pos] = make_float4(pos.x, pos.y, pos.z, flag);
        if (d_tdb != NULL)
            d_tdb[write_pos] = make_float4(type, diameter, body, 0.0f);
        if (d_cell_orientation != NULL)
            d_cell_orientation[write_pos] = orientation;
        if (d_cell_idx != NULL)
            d_cell_idx[write_pos] = idx;
        }
    else
        {
        // handle overflow
        atomicMax(&(*d_conditions).x, size+1);
        }
    }

hipError_t gpu_compute_cell_list(unsigned int *d_cell_size,
                                  float4 *d_xyzf,
                                  float4 *d_tdb,
                                  float4 *d_cell_orientation,
                                  unsigned int *d_cell_idx,
                                  uint3 *d_conditions,
                                  const float4 *d_pos,
                                  const float4 *d_orientation,
                                  const float *d_charge,
                                  const float *d_diameter,
                                  const unsigned int *d_body,
                                  const unsigned int N,
                                  const unsigned int n_ghost,
                                  const unsigned int Nmax,
                                  const bool flag_charge,
                                  const bool flag_type,
                                  const BoxDim& box,
                                  const Index3D& ci,
                                  const Index2D& cli,
                                  const Scalar3& ghost_width)
    {
    unsigned int block_size = 256;
    int n_blocks = (int)ceil(float(N+n_ghost)/(float)block_size);
    
    hipError_t err;
    err = hipMemset(d_cell_size, 0, sizeof(unsigned int)*ci.getNumElements());
    
    if (err != hipSuccess)
        return err;
    
    gpu_compute_cell_list_kernel<<<n_blocks, block_size>>>(d_cell_size,
                                                           d_xyzf,
                                                           d_tdb,
                                                           d_cell_orientation,
                                                           d_cell_idx,
                                                           d_conditions,
                                                           d_pos,
                                                           d_orientation,
                                                           d_charge,
                                                           d_diameter,
                                                           d_body,
                                                           N,
                                                           n_ghost,
                                                           Nmax,
                                                           flag_charge,
                                                           flag_type,
                                                           box,
                                                           ci,
                                                           cli,
                                                           ghost_width);
    
    return hipSuccess;
    }

// ********************* Following are helper functions, structs, etc for the 1x optimized cell list build
//! \internal
/*! \param a First element
    \param b Second element
    The two elements are swapped
*/
template<class T> __device__ inline void swap(T & a, T & b)
    {
    T tmp = a;
    a = b;
    b = tmp;
    }

//! \internal
/*! \param shared Pointer to shared memory to bitonic sort
*/
template<class T, unsigned int block_size> __device__ inline void bitonic_sort(T *shared)
    {
    unsigned int tid = threadIdx.x;
    
    // Parallel bitonic sort.
    for (int k = 2; k <= block_size; k *= 2)
        {
        // Bitonic merge:
        for (int j = k / 2; j>0; j /= 2)
            {
            int ixj = tid ^ j;
            
            if (ixj > tid)
                {
                if ((tid & k) == 0)
                    {
                    if (shared[tid] > shared[ixj])
                        {
                        swap(shared[tid], shared[ixj]);
                        }
                    }
                else
                    {
                    if (shared[tid] < shared[ixj])
                        {
                        swap(shared[tid], shared[ixj]);
                        }
                    }
                }
                
            __syncthreads();
            }
        }
    }

//! \internal
/*! \brief Pair a particle and its assigned bin together for sorting
*/
struct bin_id_pair
    {
    unsigned int bin;   //!< Cell index
    unsigned int id;    //!< Particle id
    unsigned int start_offset;  //!< Write offset
    };

//! \internal
/*! \param bin Cell index
    \param id Particle id
*/
__device__ inline bin_id_pair make_bin_id_pair(unsigned int bin, unsigned int id)
    {
    bin_id_pair res;
    res.bin = bin;
    res.id = id;
    res.start_offset = 0;
    return res;
    }

//! \internal
/*! \param a First element
    \param b Second element
*/
__device__ inline bool operator< (const bin_id_pair& a, const bin_id_pair& b)
    {
    if (a.bin == b.bin)
        return (a.id < b.id);
    else
        return (a.bin < b.bin);
    }

//! \internal
/*! \param a First element
    \param b Second element
*/
__device__ inline bool operator> (const bin_id_pair& a, const bin_id_pair& b)
    {
    if (a.bin == b.bin)
        return (a.id > b.id);
    else
        return (a.bin > b.bin);
    }

//! \internal
/*! \param temp Temporary array in shared memory to scan
*/
template<class T, unsigned int block_size> __device__ inline void scan_naive(T *temp)
    {
    int thid = threadIdx.x;
    
    int pout = 0;
    int pin = 1;
    
    for (int offset = 1; offset < block_size; offset *= 2)
        {
        pout = 1 - pout;
        pin  = 1 - pout;
        __syncthreads();
        
        temp[pout*block_size+thid] = temp[pin*block_size+thid];
        
        if (thid >= offset)
            temp[pout*block_size+thid] += temp[pin*block_size+thid - offset];
        }
        
    __syncthreads();
    // bring the data back to the initial array
    if (pout == 1)
        {
        pout = 1 - pout;
        pin  = 1 - pout;
        temp[pout*block_size+thid] = temp[pin*block_size+thid];
        __syncthreads();
        }
    }

//! Kernel that computes the cell list on the GPU
/*! \param d_cell_size Number of particles in each cell
    \param d_xyzf Cell XYZF data array
    \param d_tdb Cell TDB data array
    \param d_cell_orientation Particle orientation in cell list
    \param d_cell_idx Particle index in cell list
    \param d_conditions Conditions flags for detecting overflow and other error conditions
    \param d_pos Particle position array
    \param d_orientation Particle orientation array
    \param d_charge Particle charge array
    \param d_diameter Particle diameter array
    \param d_body Particle body array
    \param N Number of particles
    \param n_ghost Number of ghost particles
    \param Nmax Maximum number of particles that can be placed in a single cell
    \param flag_charge Set to true to store chage in the flag position in \a d_xyzf
    \param flag_type Set to true to store type in the flag position in \a d_xyzf
    \param box Box dimensions
    \param ci Indexer to compute cell id from cell grid coords
    \param cli Indexer to index into \a d_xyzf and \a d_tdb
    \param ghost_width width of ghost layer
    
    \note Optimized for compute 1.x hardware
*/
template<unsigned int block_size>
__global__ void gpu_compute_cell_list_1x_kernel(unsigned int *d_cell_size,
                                                float4 *d_xyzf,
                                                float4 *d_tdb,
                                                float4 *d_cell_orientation,
                                                unsigned int *d_cell_idx,
                                                uint3 *d_conditions,
                                                const float4 *d_pos,
                                                const float4 *d_orientation,
                                                const float *d_charge,
                                                const float *d_diameter,
                                                const unsigned int *d_body,
                                                const unsigned int N,
                                                const unsigned int n_ghost,
                                                const unsigned int Nmax,
                                                const bool flag_charge,
                                                const bool flag_type,
                                                const BoxDim box,
                                                const Index3D ci,
                                                const Index2D cli,
                                                const Scalar3 ghost_width)
    {
    // sentinel to label a bin as invalid
    const unsigned int INVALID_BIN = 0xffffffff;

    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    float4 postype = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    if (idx < N + n_ghost)
        postype = d_pos[idx];

    float3 pos = make_float3(postype.x, postype.y, postype.z);

    // find the bin each particle belongs in
    Scalar3 f = box.makeFraction(pos,ghost_width);
    
    unsigned int ib = (unsigned int)(f.x * ci.getW());
    unsigned int jb = (unsigned int)(f.y * ci.getH());
    unsigned int kb = (unsigned int)(f.z * ci.getD());
    
    uchar3 periodic = box.getPeriodic();

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;
        
    unsigned int bin = ci(ib, jb, kb);

    // check if the particle is inside the unit cell + ghost layer
    if ((!periodic.x && (f.x < Scalar(0.0) || f.x >= Scalar(1.0))) ||
        (!periodic.y && (f.y < Scalar(0.0) || f.y >= Scalar(1.0))) ||
        (!periodic.z && (f.z < Scalar(0.0) || f.z >= Scalar(1.0))) )
        {
        // silently ignore ghost particles that are outside the dimensions
        if (idx < N) (*d_conditions).z = idx+1;
        bin = INVALID_BIN;
        }

    // check for nan pos
    if (isnan(pos.x) || isnan(pos.y) || isnan(pos.z))
        {
        (*d_conditions).y = idx+1;
        bin = INVALID_BIN;
        }

    // if we are past the end of the array, mark the bin as invalid
    if (idx >= N + n_ghost)
        bin = INVALID_BIN;


    // now, perform the fun sorting and bin entry
    // load up shared memory
    __shared__ bin_id_pair bin_pairs[block_size];
    bin_pairs[threadIdx.x] = make_bin_id_pair(bin, idx);
    __syncthreads();
    
    // sort it
    bitonic_sort<bin_id_pair, block_size>(bin_pairs);
    
    // identify the breaking points
    __shared__ unsigned int unique[block_size*2+1];
    
    bool is_unique = false;
    if (threadIdx.x > 0 && bin_pairs[threadIdx.x].bin != bin_pairs[threadIdx.x-1].bin)
        is_unique = true;
        
    unique[threadIdx.x] = 0;
    if (is_unique)
        unique[threadIdx.x] = 1;
        
    // threadIdx.x = 0 is unique: but we don't want to count it in the scan
    if (threadIdx.x == 0)
        is_unique = true;
        
    __syncthreads();
    
    // scan to find addresses to write to
    scan_naive<unsigned int, block_size>(unique);
    
    // determine start location of each unique value in the array
    // save shared memory by reusing the temp data in the unique[] array
    unsigned int *start = &unique[block_size];
    
    if (is_unique)
        start[unique[threadIdx.x]] = threadIdx.x;
        
    // boundary condition: need one past the end
    if (threadIdx.x == 0)
        start[unique[block_size-1]+1] = block_size;
        
    __syncthreads();
    
    bool is_valid = (bin_pairs[threadIdx.x].bin < ci.getNumElements());
    
    // now: each unique start point does it's own atomicAdd to find the starting offset
    // the is_valid check is to prevent writing to out of bounds memory at the tail end of the array
    if (is_unique && is_valid)
        bin_pairs[unique[threadIdx.x]].start_offset = atomicAdd(&d_cell_size[bin_pairs[threadIdx.x].bin], start[unique[threadIdx.x]+1] - start[unique[threadIdx.x]]);
        
    __syncthreads();
    
    // finally! we can write out all the particles
    // the is_valid check is to prevent writing to out of bounds memory at the tail end of the array
    unsigned int offset = bin_pairs[unique[threadIdx.x]].start_offset;
    unsigned int size = offset + threadIdx.x - start[unique[threadIdx.x]];
    if (size < Nmax)
        {
        if (is_valid)
            {
            unsigned int write_id = bin_pairs[threadIdx.x].id;
            unsigned int write_location = cli(size, bin_pairs[threadIdx.x].bin);
            
            float4 write_pos = d_pos[write_id];
            float flag = 0.0f;
            float diameter = 0.0f;
            float body = 0;
            float type = write_pos.w;
            float4 orientation = make_float4(0,0,0,0);
            if (d_tdb != NULL)
                {
                diameter = d_diameter[write_id];
                body = __int_as_float(d_body[write_id]);
                }
            if (d_cell_orientation != NULL)
                {
                orientation = d_orientation[write_id];
                }
            
            if (flag_charge)
                flag = d_charge[write_id];
            else if (flag_type)
                flag = type;
            else
                flag = __int_as_float(write_id);
            
            d_xyzf[write_location] = make_float4(write_pos.x, write_pos.y, write_pos.z, flag);
            if (d_tdb != NULL)
                d_tdb[write_location] = make_float4(type, diameter, body, 0.0f);
            if (d_cell_orientation != NULL)
                d_cell_orientation[write_location] = orientation;
            if (d_cell_idx != NULL)
                d_cell_idx[write_location] = write_id;
            }
        }
    else
        {
        // handle overflow
        atomicMax(&(*d_conditions).x, size+1);
        }
    }

hipError_t gpu_compute_cell_list_1x(unsigned int *d_cell_size,
                                     float4 *d_xyzf,
                                     float4 *d_tdb,
                                     float4 *d_cell_orientation,
                                     unsigned int *d_cell_idx,
                                     uint3 *d_conditions,
                                     const float4 *d_pos,
                                     const float4 *d_orientation,
                                     const float *d_charge,
                                     const float *d_diameter,
                                     const unsigned int *d_body,
                                     const unsigned int N,
                                     const unsigned int n_ghost,
                                     const unsigned int Nmax,
                                     const bool flag_charge,
                                     const bool flag_type,
                                     const BoxDim& box,
                                     const Index3D& ci,
                                     const Index2D& cli,
                                     const Scalar3& ghost_width)
    {
    const unsigned int block_size = 64;
    int n_blocks = (int)ceil(float(N+n_ghost)/(float)block_size);
    
    hipError_t err;
    err = hipMemset(d_cell_size, 0, sizeof(unsigned int)*ci.getNumElements());
    
    if (err != hipSuccess)
        return err;
    
    gpu_compute_cell_list_1x_kernel<block_size>
                                   <<<n_blocks, block_size>>>(d_cell_size,
                                                              d_xyzf,
                                                              d_tdb,
                                                              d_cell_orientation,
                                                              d_cell_idx,
                                                              d_conditions,
                                                              d_pos,
                                                              d_orientation,
                                                              d_charge,
                                                              d_diameter,
                                                              d_body,
                                                              N,
                                                              n_ghost,
                                                              Nmax,
                                                              flag_charge,
                                                              flag_type,
                                                              box,
                                                              ci,
                                                              cli,
                                                              ghost_width);
    
    return hipSuccess;
    }

