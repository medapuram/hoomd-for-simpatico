#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "NeighborListGPUBinned.cuh"

/*! \file NeighborListGPUBinned.cu
    \brief Defines GPU kernel code for O(N) neighbor list generation on the GPU
*/

//! Texture for reading d_cell_xyzf
texture<float4, 1, hipReadModeElementType> cell_xyzf_1d_tex;

//! Kernel call for generating neighbor list on the GPU
/*! \tparam flags Set bit 1 to enable body filtering. Set bit 2 to enable diameter filtering. 
    \param d_nlist Neighbor list data structure to write
    \param d_n_neigh Number of neighbors to write
    \param d_last_updated_pos Particle positions at this update are written to this array
    \param d_conditions Conditions array for writing overflow condition
    \param nli Indexer to access \a d_nlist
    \param d_pos Particle positions
    \param d_body Particle body indices
    \param d_diameter Particle diameters
    \param N Number of particles
    \param d_cell_size Number of particles in each cell
    \param d_cell_xyzf Cell contents (xyzf array from CellList with flag=type)
    \param d_cell_tdb Cell contents (tdb array from CellList with)
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer for indexing cells
    \param cli Cell list indexer for indexing into d_cell_xyzf
    \param cadji Adjacent cell indexer listing the 27 neighboring cells
    \param box Simulation box dimensions
    \param r_maxsq The maximum radius for which to include particles as neighbors, squared
    \param r_max The maximum radius for which to include particles as neighbors
    \param ghost_width Width of ghost cell layer
    
    \note optimized for Fermi
*/
template<unsigned char flags>
__global__ void gpu_compute_nlist_binned_new_kernel(unsigned int *d_nlist,
                                                    unsigned int *d_n_neigh,
                                                    float4 *d_last_updated_pos,
                                                    unsigned int *d_conditions,
                                                    const Index2D nli,
                                                    const float4 *d_pos,
                                                    const unsigned int *d_body,
                                                    const float *d_diameter,
                                                    const unsigned int N,
                                                    const unsigned int *d_cell_size,
                                                    const float4 *d_cell_xyzf,
                                                    const float4 *d_cell_tdb,
                                                    const unsigned int *d_cell_adj,
                                                    const Index3D ci,
                                                    const Index2D cli,
                                                    const Index2D cadji,
                                                    const BoxDim box,
                                                    const float r_maxsq,
                                                    const float r_max,
                                                    const Scalar3 ghost_width) 
    {
    bool filter_body = flags & 1;
    bool filter_diameter = flags & 2;

    // each thread is going to compute the neighbor list for a single particle
    int my_pidx = blockDim.x * blockIdx.x + threadIdx.x;

    // track the number of neighbors needed
    unsigned int n_neigh_needed = 0;

    // quit early if we are past the end of the array
    if (my_pidx >= N)
        return;

    // first, determine which bin this particle belongs to
    float4 my_postype = d_pos[my_pidx];
    float3 my_pos = make_float3(my_postype.x, my_postype.y, my_postype.z);

    unsigned int my_body = d_body[my_pidx];
    float my_diameter = d_diameter[my_pidx];

    Scalar3 f = box.makeFraction(my_pos, ghost_width);

    // find the bin each particle belongs in
    int ib = (int)(f.x * ci.getW());
    int jb = (int)(f.y * ci.getH());
    int kb = (int)(f.z * ci.getD());

    uchar3 periodic = box.getPeriodic();

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    int my_cell = ci(ib,jb,kb);

    // each thread will determine the neighborlist of a single particle
    // count number of neighbors found so far in n_neigh
    int n_neigh = 0;

    // loop over all adjacent bins
    for (unsigned int cur_adj = 0; cur_adj < cadji.getW(); cur_adj++)
        {
        int neigh_cell = d_cell_adj[cadji(cur_adj, my_cell)];
        unsigned int size = d_cell_size[neigh_cell];
    
        // now, we are set to loop through the array
        for (int cur_offset = 0; cur_offset < size; cur_offset++)
            {
            float4 cur_xyzf = tex1Dfetch(cell_xyzf_1d_tex, cli(cur_offset, neigh_cell));
            float4 cur_tdb = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            if (filter_diameter || filter_body)
                cur_tdb = d_cell_tdb[cli(cur_offset, neigh_cell)];
            unsigned int neigh_body = __float_as_int(cur_tdb.z);
            float neigh_diameter = cur_tdb.y;

            float3 neigh_pos = make_float3(cur_xyzf.x,
                                           cur_xyzf.y,
                                           cur_xyzf.z);
            int cur_neigh = __float_as_int(cur_xyzf.w);

            // compute the distance between the two particles
            float3 dx = my_pos - neigh_pos;

            // wrap the periodic boundary conditions
            dx = box.minImage(dx);

            // compute dr squared
            float drsq = dot(dx,dx);

            bool excluded = (my_pidx == cur_neigh);
 
            if (filter_body && my_body != 0xffffffff)
                excluded = excluded | (my_body == neigh_body);

            float sqshift = 0.0f;
            if (filter_diameter)
                {
                // compute the shift in radius to accept neighbors based on their diameters
                float delta = (my_diameter + neigh_diameter) * 0.5f - 1.0f;
                // r^2 < (r_max + delta)^2
                // r^2 < r_maxsq + delta^2 + 2*r_max*delta
                sqshift = (delta + 2.0f * r_max) * delta;
                }

            if (drsq <= (r_maxsq + sqshift) && !excluded)
                {
                // regular particle
                if (n_neigh < nli.getH())
                    d_nlist[nli(my_pidx, n_neigh)] = cur_neigh;
                else
                    n_neigh_needed = n_neigh+1;

                n_neigh++;
                }
            }
        }

    d_n_neigh[my_pidx] = n_neigh;

    d_last_updated_pos[my_pidx] = my_postype;

    if (n_neigh_needed > 0)
        atomicMax(&d_conditions[0], n_neigh_needed);
    }

hipError_t gpu_compute_nlist_binned(unsigned int *d_nlist,
                                     unsigned int *d_n_neigh,
                                     float4 *d_last_updated_pos,
                                     unsigned int *d_conditions,
                                     const Index2D& nli,
                                     const float4 *d_pos,
                                     const unsigned int *d_body,
                                     const float *d_diameter,
                                     const unsigned int N,
                                     const unsigned int *d_cell_size,
                                     const float4 *d_cell_xyzf,
                                     const float4 *d_cell_tdb,
                                     const unsigned int *d_cell_adj,
                                     const Index3D& ci,
                                     const Index2D& cli,
                                     const Index2D& cadji,
                                     const BoxDim& box,
                                     const float r_maxsq,
                                     const unsigned int block_size,
                                     bool filter_body,
                                     bool filter_diameter,
                                     const Scalar3& ghost_width)
    {
    int n_blocks = (int)ceil(float(N)/(float)block_size);

    // bind the position texture
    cell_xyzf_1d_tex.normalized = false;
    cell_xyzf_1d_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, cell_xyzf_1d_tex, d_cell_xyzf, sizeof(Scalar4)*(cli.getNumElements()));
    if (error != hipSuccess)
        return error;


    if (!filter_diameter && !filter_body)
        {
        gpu_compute_nlist_binned_new_kernel<0><<<n_blocks, block_size>>>(d_nlist,
                                                                         d_n_neigh,
                                                                         d_last_updated_pos,
                                                                         d_conditions,
                                                                         nli,
                                                                         d_pos,
                                                                         d_body,
                                                                         d_diameter,
                                                                         N,
                                                                         d_cell_size,
                                                                         d_cell_xyzf,
                                                                         d_cell_tdb,
                                                                         d_cell_adj,
                                                                         ci,
                                                                         cli,
                                                                         cadji,
                                                                         box,
                                                                         r_maxsq,
                                                                         sqrtf(r_maxsq),
                                                                         ghost_width);
        }
    if (!filter_diameter && filter_body)
        {
        gpu_compute_nlist_binned_new_kernel<1><<<n_blocks, block_size>>>(d_nlist,
                                                                         d_n_neigh,
                                                                         d_last_updated_pos,
                                                                         d_conditions,
                                                                         nli,
                                                                         d_pos,
                                                                         d_body,
                                                                         d_diameter,
                                                                         N,
                                                                         d_cell_size,
                                                                         d_cell_xyzf,
                                                                         d_cell_tdb,
                                                                         d_cell_adj,
                                                                         ci,
                                                                         cli,
                                                                         cadji,
                                                                         box,
                                                                         r_maxsq,
                                                                         sqrtf(r_maxsq),
                                                                         ghost_width);
        }
    if (filter_diameter && !filter_body)
        {
        gpu_compute_nlist_binned_new_kernel<2><<<n_blocks, block_size>>>(d_nlist,
                                                                         d_n_neigh,
                                                                         d_last_updated_pos,
                                                                         d_conditions,
                                                                         nli,
                                                                         d_pos,
                                                                         d_body,
                                                                         d_diameter,
                                                                         N,
                                                                         d_cell_size,
                                                                         d_cell_xyzf,
                                                                         d_cell_tdb,
                                                                         d_cell_adj,
                                                                         ci,
                                                                         cli,
                                                                         cadji,
                                                                         box,
                                                                         r_maxsq,
                                                                         sqrtf(r_maxsq),
                                                                         ghost_width);
        }
    if (filter_diameter && filter_body)
        {
        gpu_compute_nlist_binned_new_kernel<3><<<n_blocks, block_size>>>(d_nlist,
                                                                         d_n_neigh,
                                                                         d_last_updated_pos,
                                                                         d_conditions,
                                                                         nli,
                                                                         d_pos,
                                                                         d_body,
                                                                         d_diameter,
                                                                         N,
                                                                         d_cell_size,
                                                                         d_cell_xyzf,
                                                                         d_cell_tdb,
                                                                         d_cell_adj,
                                                                         ci,
                                                                         cli,
                                                                         cadji,
                                                                         box,
                                                                         r_maxsq,
                                                                         sqrtf(r_maxsq),
                                                                         ghost_width);
        }


    return hipSuccess;
    }

//! Texture for reading d_cell_adj
texture<unsigned int, 2, hipReadModeElementType> cell_adj_tex;
//! Texture for reading d_cell_size
texture<unsigned int, 1, hipReadModeElementType> cell_size_tex;
//! Texture for reading d_cell_xyzf
texture<float4, 2, hipReadModeElementType> cell_xyzf_tex;
//! Texture for reading d_cell_tdb
texture<float4, 2, hipReadModeElementType> cell_tdb_tex;

//! Kernel call for generating neighbor list on the GPU
/*! \tparam filter_flags Set bit 1 to enable body filtering. Set bit 2 to enable diameter filtering.
    \param d_nlist Neighbor list data structure to write
    \param d_n_neigh Number of neighbors to write
    \param d_last_updated_pos Particle positions at this update are written to this array
    \param d_conditions Conditions array for writing overflow condition
    \param nli Indexer to access \a d_nlist
    \param d_pos Particle positions
    \param d_body Particle body indices
    \param d_diameter Particle diameters
    \param N Number of particles
    \param ci Cell indexer for indexing cells
    \param box Simulation box dimensions
    \param r_maxsq The maximum radius for which to include particles as neighbors, squared
    \param r_max The maximum radius for which to include particles as neighbors
    \param ghost_width Width of ghost cell layer
    
    \note optimized for compute 1.x devices
*/
template<unsigned char filter_flags>
__global__ void gpu_compute_nlist_binned_1x_kernel(unsigned int *d_nlist,
                                                   unsigned int *d_n_neigh,
                                                   float4 *d_last_updated_pos,
                                                   unsigned int *d_conditions,
                                                   const Index2D nli,
                                                   const float4 *d_pos,
                                                   const unsigned int *d_body,
                                                   const float *d_diameter,
                                                   const unsigned int N,
                                                   const Index3D ci,
                                                   const BoxDim box,
                                                   const float r_maxsq,
                                                   const float r_max,
                                                   const Scalar3 ghost_width)
    {
    bool filter_body = filter_flags & 1;
    bool filter_diameter = filter_flags & 2;

    // each thread is going to compute the neighbor list for a single particle
    int my_pidx = blockDim.x * blockIdx.x + threadIdx.x;

    // count the number of neighbors needed
    unsigned int n_neigh_needed = 0;

    // quit early if we are past the end of the array
    if (my_pidx >= N)
        return;

    // first, determine which bin this particle belongs to
    float4 my_postype = d_pos[my_pidx];
    float3 my_pos = make_float3(my_postype.x, my_postype.y, my_postype.z);

    unsigned int my_body = d_body[my_pidx];
    float my_diameter = d_diameter[my_pidx];

    // get periodic flags
    uchar3 periodic = box.getPeriodic();

    // find the bin each particle belongs in
    Scalar3 f = box.makeFraction(my_pos,ghost_width);
    unsigned int ib = (unsigned int)(f.x * ci.getW());
    unsigned int jb = (unsigned int)(f.y * ci.getH());
    unsigned int kb = (unsigned int)(f.z * ci.getD());

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    int my_cell = ci(ib,jb,kb);

    // each thread will determine the neighborlist of a single particle
    // count number of neighbors found so far in n_neigh
    int n_neigh = 0;

    // loop over all adjacent bins
    for (unsigned int cur_adj = 0; cur_adj < 27; cur_adj++)
        {
        int neigh_cell = tex2D(cell_adj_tex, cur_adj, my_cell);
        unsigned int size = tex1Dfetch(cell_size_tex, neigh_cell);

        float4 next_xyzf = tex2D(cell_xyzf_tex, 0, neigh_cell);

        // now, we are set to loop through the array
        for (int cur_offset = 0; cur_offset < size; cur_offset++)
            {
            float4 cur_xyzf = next_xyzf;
            next_xyzf = tex2D(cell_xyzf_tex, cur_offset+1, neigh_cell);
            float4 cur_tdb = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            if (filter_diameter || filter_body)
                cur_tdb = tex2D(cell_tdb_tex, cur_offset, neigh_cell);
            unsigned int neigh_body = __float_as_int(cur_tdb.z);
            float neigh_diameter = cur_tdb.y;

            float3 neigh_pos = make_float3(cur_xyzf.x,
                                           cur_xyzf.y,
                                           cur_xyzf.z);
            int cur_neigh = __float_as_int(cur_xyzf.w);

            // compute the distance between the two particles
            float3 dx = my_pos - neigh_pos;

            // wrap the periodic boundary conditions
            dx = box.minImage(dx);
            // compute dr squared
            float drsq = dot(dx,dx);


            bool excluded = (my_pidx == cur_neigh);

            if (filter_body && my_body != 0xffffffff)
                excluded = excluded | (my_body == neigh_body);

            float sqshift = 0.0f;
            if (filter_diameter)
                {
                // compute the shift in radius to accept neighbors based on their diameters
                float delta = (my_diameter + neigh_diameter) * 0.5f - 1.0f;
                // r^2 < (r_max + delta)^2
                // r^2 < r_maxsq + delta^2 + 2*r_max*delta
                sqshift = (delta + 2.0f * r_max) * delta;
                }

            if (drsq <= (r_maxsq + sqshift) && !excluded)
                {
                if (n_neigh < nli.getH())
                    d_nlist[nli(my_pidx, n_neigh)] = cur_neigh;
                else
                    n_neigh_needed = n_neigh+1;

                n_neigh++;
                }
            }
        }

    d_n_neigh[my_pidx] = n_neigh;
    d_last_updated_pos[my_pidx] = my_postype;

    if (n_neigh_needed > 0)
        atomicMax(&d_conditions[0], n_neigh_needed);
    }

hipError_t gpu_compute_nlist_binned_1x(unsigned int *d_nlist,
                                        unsigned int *d_n_neigh,
                                        float4 *d_last_updated_pos,
                                        unsigned int *d_conditions,
                                        const Index2D& nli,
                                        const float4 *d_pos,
                                        const unsigned int *d_body,
                                        const float *d_diameter,
                                        const unsigned int N,
                                        const unsigned int *d_cell_size,
                                        const hipArray *dca_cell_xyzf,
                                        const hipArray *dca_cell_tdb,
                                        const hipArray *dca_cell_adj,
                                        const Index3D& ci,
                                        const BoxDim& box,
                                        const float r_maxsq,
                                        const unsigned int block_size,
                                        bool filter_body,
                                        bool filter_diameter,
                                        const Scalar3& ghost_width)
    {
    int n_blocks = (int)ceil(float(N)/(float)block_size);
    
    hipError_t err = hipBindTextureToArray(cell_adj_tex, dca_cell_adj);
    if (err != hipSuccess)
        return err;
    
    err = hipBindTextureToArray(cell_xyzf_tex, dca_cell_xyzf);
    if (err != hipSuccess)
        return err;
    
    err = hipBindTextureToArray(cell_tdb_tex, dca_cell_tdb);
    if (err != hipSuccess)
        return err;
    
    err = hipBindTexture(0, cell_size_tex, d_cell_size, sizeof(unsigned int)*ci.getNumElements());
    if (err != hipSuccess)
        return err;

    if (!filter_diameter && !filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<0><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width);
        }
    if (!filter_diameter && filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<1><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width);
        }
    if (filter_diameter && !filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<2><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width);
        }
    if (filter_diameter && filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<3><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width );
        }
    return hipSuccess;
    }

/*! Call this method once at initialization. It specifies that gpu_compute_nlist_binned_new_kernel() utilize the 48k
    L1 cache on Fermi.
*/
hipError_t gpu_setup_compute_nlist_binned()
    {
    hipError_t error;
    error = hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_compute_nlist_binned_new_kernel<0>), hipFuncCachePreferL1);
    if (error != hipSuccess)
        return error;
    error = hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_compute_nlist_binned_new_kernel<1>), hipFuncCachePreferL1);
    if (error != hipSuccess)
        return error;
    error = hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_compute_nlist_binned_new_kernel<2>), hipFuncCachePreferL1);
    if (error != hipSuccess)
        return error;
    error = hipFuncSetCacheConfig(reinterpret_cast<const void*>(gpu_compute_nlist_binned_new_kernel<3>), hipFuncCachePreferL1);
    return error;
    }

