#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander / Everyone is free to add additional potentials

/*! \file AllDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairLJ.h"
#include "EvaluatorPairGauss.h"
#include "EvaluatorPairSLJ.h"
#include "EvaluatorPairYukawa.h"
#include "EvaluatorPairMorse.h"
#include "PotentialPairDPDThermoGPU.cuh"
#include "EvaluatorPairDPDThermo.h"
#include "AllDriverPotentialPairGPU.cuh"
#include "EvaluatorPairEwald.h"
#include "EvaluatorPairDPDLJThermo.h"
#include "EvaluatorPairForceShiftedLJ.h"

hipError_t gpu_compute_ljtemp_forces(const pair_args_t& pair_args,
                                      const float2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairLJ>(pair_args,
                                                    d_params);
    }

hipError_t gpu_compute_gauss_forces(const pair_args_t& pair_args,
                                     const float2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairGauss>(pair_args,
                                                       d_params);
    }

hipError_t gpu_compute_slj_forces(const pair_args_t& pair_args,
                                   const float2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairSLJ>(pair_args,
                                                     d_params);
    }

hipError_t gpu_compute_yukawa_forces(const pair_args_t& pair_args,
                                      const float2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairYukawa>(pair_args,
                                                        d_params);
    }


hipError_t gpu_compute_morse_forces(const pair_args_t& pair_args,
                                      const float4 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairMorse>(pair_args,
                                                       d_params);
    }

hipError_t gpu_compute_dpdthermodpd_forces(const dpd_pair_args_t& args,
                                            const float2 *d_params)
    {
    return gpu_compute_dpd_forces<EvaluatorPairDPDThermo>(args,
                                                          d_params);
    }


hipError_t gpu_compute_dpdthermo_forces(const pair_args_t& pair_args,
                                         const float2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairDPDThermo>(pair_args,
                                                           d_params);
    }


hipError_t gpu_compute_ewald_forces(const pair_args_t& pair_args,
                                     const float *d_params)
    {
    return  gpu_compute_pair_forces<EvaluatorPairEwald>(pair_args,
                                                        d_params);
    }


hipError_t gpu_compute_dpdljthermodpd_forces(const dpd_pair_args_t& args,
                                              const float4 *d_params)
    {
    return gpu_compute_dpd_forces<EvaluatorPairDPDLJThermo>(args,
                                                            d_params);
    }


hipError_t gpu_compute_dpdljthermo_forces(const pair_args_t& args,
                                           const float4 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairDPDLJThermo>(args,
                                                             d_params);
    }

hipError_t gpu_compute_force_shifted_lj_forces(const pair_args_t & args,
                                                const float2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairForceShiftedLJ>(args,
                                                                d_params);
    }
