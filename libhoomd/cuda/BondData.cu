#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "BondData.cuh"
#include "ParticleData.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file BondData.cu
    \brief Implements the helper functions (GPU version) for updating the GPU bond table
*/

#define MAX(i,j) (i > j ? i : j)

//! Kernel to find the maximum number of angles per particle
__global__ void gpu_find_max_bond_number_kernel(const uint2 *bonds,
                                             const unsigned int *d_rtag,
                                             unsigned int *d_n_bonds,
                                             unsigned int num_bonds,
                                             unsigned int N,
                                             unsigned int n_ghosts,
                                             const unsigned int cur_max,
                                             unsigned int *condition)
    {
    int bond_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bond_idx >= num_bonds)
        return;

    uint2 bond = bonds[bond_idx];
    unsigned int tag1 = bond.x;
    unsigned int tag2 = bond.y;
    unsigned int idx1 = d_rtag[tag1];
    unsigned int idx2 = d_rtag[tag2];

    bool bond_needed = false;
    bool bond_valid = true;
    if (idx1 < N)
        {
        unsigned int n = atomicInc(&d_n_bonds[idx1], 0xffffffff);
        bond_valid &= (idx2 < N + n_ghosts);
        if (n >= cur_max) bond_needed = true;
        }
    if (idx2 < N)
        {
        unsigned int n = atomicInc(&d_n_bonds[idx2], 0xffffffff);
        bond_valid &= (idx1 < N + n_ghosts);
        if (n >= cur_max) bond_needed = true;
        }

    if (bond_needed)
        atomicOr(condition, 1);
    if (!bond_valid)
        atomicOr(condition, 2);
    }

//! Kernel to fill the GPU bond table
__global__ void gpu_fill_gpu_bond_table(const uint2 *bonds,
                                        const unsigned int *bond_type,
                                        uint2 *gpu_btable,
                                        const unsigned int pitch,
                                        const unsigned int *d_rtag,
                                        unsigned int *d_n_bonds,
                                        unsigned int num_bonds,
                                        unsigned int N)
    {
    int bond_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (bond_idx >= num_bonds)
        return;

    uint2 bond = bonds[bond_idx];
    unsigned int tag1 = bond.x;
    unsigned int tag2 = bond.y;
    unsigned int type = bond_type[bond_idx];
    unsigned int idx1 = d_rtag[tag1];
    unsigned int idx2 = d_rtag[tag2];

    if (idx1 < N)
        {
        unsigned int num1 = atomicInc(&d_n_bonds[idx1],0xffffffff);
        gpu_btable[num1*pitch+idx1] = make_uint2(idx2,type);
        }
    if (idx2 < N)
        {
        unsigned int num2 = atomicInc(&d_n_bonds[idx2],0xffffffff);
        gpu_btable[num2*pitch+idx2] = make_uint2(idx1,type);
        }
    }


//! Find the maximum number of bonds per particle
/*! \param d_n_bonds Number of bonds per particle (return array)
    \param d_bonds Array of bonds
    \param num_bonds Size of bond array
    \param N Number of particles in the system
    \param d_rtag Array of reverse-lookup particle tag . particle index
    \param cur_max Current maximum bonded particle number
    \param d_condition Condition variable, set to unequal zero if we exceed the maximum numbers
 */
hipError_t gpu_find_max_bond_number(unsigned int *d_n_bonds,
                                     const uint2 *d_bonds,
                                     const unsigned int num_bonds,
                                     const unsigned int N,
                                     const unsigned int n_ghosts,
                                     const unsigned int *d_rtag,
                                     const unsigned int cur_max,
                                     unsigned int *d_condition)
    {
    assert(d_bonds);
    assert(d_rtag);
    assert(d_n_bonds);

    unsigned int block_size = 512;

    // clear n_bonds array
    hipMemset(d_n_bonds, 0, sizeof(unsigned int) * N);

    gpu_find_max_bond_number_kernel<<<num_bonds/block_size + 1, block_size>>>(d_bonds,
                                                                              d_rtag,
                                                                              d_n_bonds,
                                                                              num_bonds,
                                                                              N,
                                                                              n_ghosts,
                                                                              cur_max,
                                                                              d_condition);

    return hipSuccess;
    }

//! Construct the GPU bond table
/*! \param d_gpu_bondtable Pointer to the bond table on the GPU
    \param d_n_bonds Number of bonds per particle (return array)
    \param d_bonds Bonds array
    \param d_bond_type Array of bond types
    \param d_rtag Reverse-lookup tag->index
    \param num_bonds Number of bonds in bond list
    \param pitch Pitch of 2D bondtable array
    \param N Number of particles
 */
hipError_t gpu_create_bondtable(uint2 *d_gpu_bondtable,
                                 unsigned int *d_n_bonds,
                                 const uint2 *d_bonds,
                                 const unsigned int *d_bond_type,
                                 const unsigned int *d_rtag,
                                 const unsigned int num_bonds,
                                 unsigned int pitch,
                                 unsigned int N)
    {
    unsigned int block_size = 512;

    // clear n_bonds array
    hipMemset(d_n_bonds, 0, sizeof(unsigned int) * N);

    gpu_fill_gpu_bond_table<<<num_bonds/block_size + 1, block_size>>>(d_bonds,
                                                                      d_bond_type,
                                                                      d_gpu_bondtable,
                                                                      pitch,
                                                                      d_rtag,
                                                                      d_n_bonds,
                                                                      num_bonds,
                                                                      N);
    return hipSuccess;
    }

//! Kernel to mark duplicate received bonds
__global__ void gpu_mark_recv_bond_duplicates_kernel(const unsigned int n_bonds,
                                         const bond_element *recv_bonds,
                                         unsigned int *bond_remove_mask,
                                         const unsigned int n_recv_bonds,
                                         unsigned int *bond_rtag,
                                         unsigned char *recv_bond_active,
                                         unsigned int *n_duplicate_recv_bonds)
    {
    unsigned int recv_idx = blockIdx.x *blockDim.x + threadIdx.x;

    if (recv_idx >= n_recv_bonds) return;

    const bond_element& el = recv_bonds[recv_idx];
    unsigned int tag = el.tag;
   
    // stage the bond
    unsigned int rtag = atomicMin(&bond_rtag[tag], (unsigned int) BOND_NOT_LOCAL-1);

    bool duplicate = false;

    if (rtag != BOND_NOT_LOCAL)
        {
        bool remove = false;
        if (rtag < n_bonds)
            remove = bond_remove_mask[rtag];

        // if the bond is a duplicate of a local bond which is not removed, mark it
        if (! remove)
            {
            duplicate = true;
            atomicInc(n_duplicate_recv_bonds, 0xffffffff);
            }
        }

    recv_bond_active[recv_idx] = duplicate ? 0 : 1;
    }

//! Mark duplicate bonds received
/*! \param n_bonds Number of bonds in local bond table
    \param d_recv_bonds Buffer of received bonds
    \param d_bond_remove_mask Flags for every local bond to indicate removal
    \param n_recv_bonds Number of bonds received
    \param d_bond_rtag Bond tag->idx lookup
    \param d_recv_bond_active Per-received bond flag, 1 if unique, 0 if duplicate (return values)
    \param d_n_duplicate_recv_bonds Number of duplicates found (return value)
 */
void gpu_mark_recv_bond_duplicates(const unsigned int n_bonds,
                                   const bond_element *d_recv_bonds,
                                   unsigned int *d_bond_remove_mask,
                                   const unsigned int n_recv_bonds,
                                   unsigned int *d_bond_rtag,
                                   unsigned char *d_recv_bond_active,
                                   unsigned int *d_n_duplicate_recv_bonds)
    {
    hipMemsetAsync(d_n_duplicate_recv_bonds, 0, sizeof(unsigned int));

    unsigned int block_size = 512;

    gpu_mark_recv_bond_duplicates_kernel<<<n_recv_bonds/block_size+1,block_size>>>(
        n_bonds,
        d_recv_bonds,
        d_bond_remove_mask,
        n_recv_bonds,
        d_bond_rtag,
        d_recv_bond_active,
        d_n_duplicate_recv_bonds);
    }

//! Kernel to backfill the local bond table with received bonds and remove non-local bonds
__global__ void gpu_fill_bondtable_kernel(const unsigned int old_n_bonds,
                                               const unsigned int n_recv_bonds,
                                               const unsigned int n_unique_recv_bonds,
                                               const unsigned int n_remove_bonds,
                                               const unsigned int *remove_mask,
                                               const unsigned char *recv_bond_active,
                                               const bond_element *recv_buf,
                                               uint2 *bonds,
                                               unsigned int *bond_type,
                                               unsigned int *bond_tag,
                                               unsigned int *bond_rtag,
                                               unsigned int *n_fetch_bond)
    {
    unsigned int bond_idx = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned int new_nbonds = old_n_bonds - n_remove_bonds + n_unique_recv_bonds;
    
    if (bond_idx >= old_n_bonds + n_unique_recv_bonds) return;

    bool replace = true;

    if (bond_idx < old_n_bonds)
        {
        replace = remove_mask[bond_idx];

        // reset rtag
        if (replace) bond_rtag[bond_tag[bond_idx]] = BOND_NOT_LOCAL;
        }
    
    if (replace && bond_idx < new_nbonds)
        {
        // try to atomically fetch a bond from the received list, ignore duplicates
        bool active = false;
        unsigned int n;
        while (!active)
            {
            n = atomicInc(n_fetch_bond, 0xffffffff);
            if (n < n_recv_bonds)
                active = recv_bond_active[n];
            else
                active = true;
            }

        if (n < n_recv_bonds) 
            {
            // copy over receive buffer data
            const bond_element &el= recv_buf[n];

            bonds[bond_idx] = el.bond;
            bond_type[bond_idx] = el.type;
            bond_tag[bond_idx] = el.tag;
            }
        else
            {
            unsigned int fetch_idx = new_nbonds + (n - n_recv_bonds);
            bool remove = remove_mask[fetch_idx];

            // we should not normally read past the end of the array, if the number
            // of removed particles correctly reflects the number of remove flags set
            while (remove) {
                // reset rtags as we go
                bond_rtag[bond_tag[fetch_idx]] = BOND_NOT_LOCAL;

                n = atomicInc(n_fetch_bond, 0xffffffff);

                fetch_idx = new_nbonds + (n - n_recv_bonds);
                remove = remove_mask[fetch_idx];
                };

            // backfill with a bond from the end
            bonds[bond_idx] = bonds[fetch_idx];
            bond_type[bond_idx] = bond_type[fetch_idx];
            bond_tag[bond_idx] = bond_tag[fetch_idx];
            }
         } // if replace
    }

//! Backfill local bond table with received bonds and remove non-local bonds
/*! \param old_n_bonds Current size of bond table
    \param n_recv_bonds Size of bond receive buffer
    \param n_unique_recv_bonds Number of unique received bonds
    \param n_remove_bonds Number of bonds to be removed from local bond table
    \param d_remove_mask Flag for every bond, 1 if bond is to be removed, 0 otherwise
    \param d_recv_bond_active Flag for every received bond, 1 if unique, 0 if duplicate
    \param d_recv_buf Buffer of received bonds
    \param d_bonds Local bond table
    \param d_bond_type Local list of bond types
    \param d_bond_tag Local list of bond tags
    \param d_bond_rtag Bond tag->idx lookup table
    \param d_n_fetch_bond Temporary counter for backfilling of bonds
*/
void gpu_fill_bond_bondtable(const unsigned int old_n_bonds,
                             const unsigned int n_recv_bonds,
                             const unsigned int n_unique_recv_bonds,
                             const unsigned int n_remove_bonds,
                             const unsigned int *d_remove_mask,
                             const unsigned char *d_recv_bond_active,
                             const bond_element *d_recv_buf,
                             uint2 *d_bonds,
                             unsigned int *d_bond_type,
                             unsigned int *d_bond_tag,
                             unsigned int *d_bond_rtag,
                             unsigned int *d_n_fetch_bond)
    {
    unsigned int block_size = 512;
    
    hipMemsetAsync(d_n_fetch_bond, 0, sizeof(unsigned int));

    unsigned int end = old_n_bonds + n_unique_recv_bonds;

    gpu_fill_bondtable_kernel<<<end/block_size+1,block_size>>>(
        old_n_bonds,
        n_recv_bonds,
        n_unique_recv_bonds,
        n_remove_bonds,
        d_remove_mask,
        d_recv_bond_active,
        d_recv_buf,
        d_bonds,
        d_bond_type,
        d_bond_tag,
        d_bond_rtag,
        d_n_fetch_bond);
    }

//! Kernel to update reverse-lookup tags for bonds
__global__ void gpu_update_bond_rtags_kernel(unsigned int *bond_rtag,
                                      const unsigned int *bond_tag,
                                      const unsigned int num_bonds)
    {
    unsigned int bond_idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (bond_idx >= num_bonds) return;

    bond_rtag[bond_tag[bond_idx]] = bond_idx;
    }

//! Update the bond tag ->idx lookup table
/*! \param d_bond_rtag Reverse-lookup table
    \param d_bond_tag Local list of bond tags
    \param num_bonds Number of local bonds
 */
void gpu_update_bond_rtags(unsigned int *d_bond_rtag,
                           const unsigned int *d_bond_tag,
                           const unsigned int num_bonds)
    {
    unsigned int block_size = 512;

    gpu_update_bond_rtags_kernel<<<num_bonds/block_size+1, block_size>>>(d_bond_rtag,
                                                                         d_bond_tag,
                                                                         num_bonds);
    }
