#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "BondTablePotentialGPU.cuh"


#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file BondTablePotentialGPU.cu
    \brief Defines GPU kernel code for calculating the table bond forces. Used by BondTablePotentialGPU.
*/


//! Texture for reading table values
texture<float2, 1, hipReadModeElementType> tables_tex;

/*!  This kernel is called to calculate the table pair forces on all N particles

    \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch Pitch of 2D virial array
    \param N number of particles in system
    \param d_pos device array of particle positions
    \param box Box dimensions used to implement periodic boundary conditions
    \param blist List of bonds stored on the GPU
    \param pitch Pitch of 2D bond list
    \param n_bonds_list List of numbers of bonds stored on the GPU
    \param n_bond_type number of bond types
    \param d_params Parameters for each table associated with a type pair
    \param table_value index helper function
    \param d_flags Flag allocated on the device for use in checking for bonds that cannot be evaluated

    See BondTablePotential for information on the memory layout.

    \b Details:
    * Table entries are read from tables_tex. Note that currently this is bound to a 1D memory region. Performance tests
      at a later date may result in this changing.
*/
__global__ void gpu_compute_bondtable_forces_kernel(float4* d_force,
                                     float* d_virial,
                                     const unsigned int virial_pitch,
                                     const unsigned int N,
                                     const Scalar4 *d_pos,
                                     const BoxDim box,
                                     const uint2 *blist,
                                     const unsigned int pitch,
                                     const unsigned int *n_bonds_list,
                                     const unsigned int n_bond_type,
                                     const float4 *d_params,
                                     const Index2D table_value,
                                     unsigned int *d_flags)
    {

    
    // read in params for easy and fast access in the kernel
    extern __shared__ float4 s_params[];
    for (unsigned int cur_offset = 0; cur_offset < n_bond_type; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < n_bond_type)
            s_params[cur_offset + threadIdx.x] = d_params[cur_offset + threadIdx.x];
        }
    __syncthreads();


    // start by identifying which particle we are to handle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_bonds =n_bonds_list[idx];

    // read in the position of our particle.
    Scalar4 postype = d_pos[idx];
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);

    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    // initialize the virial tensor to 0
    float virial[6];
    for (unsigned int i = 0; i < 6; i++)
        virial[i] = 0;

    // loop over neighbors
    for (int bond_idx = 0; bond_idx < n_bonds; bond_idx++)
        {
        // MEM TRANSFER: 8 bytes
        uint2 cur_bond = blist[pitch*bond_idx + idx];

        int cur_bond_idx = cur_bond.x;
        int cur_bond_type = cur_bond.y;

        // get the bonded particle's position (MEM_TRANSFER: 16 bytes)
        Scalar4 neigh_postype = d_pos[cur_bond_idx];
        Scalar3 neigh_pos = make_scalar3(neigh_postype.x, neigh_postype.y, neigh_postype.z);

        // calculate dr (FLOPS: 3)
        float3 dx = pos - neigh_pos;

        // apply periodic boundary conditions (FLOPS: 12)
        dx = box.minImage(dx);

        // access needed parameters
        float4 params = s_params[cur_bond_type];
        float rmin = params.x;
        float rmax = params.y;
        float delta_r = params.z;

        // calculate r
        float rsq = dot(dx, dx);
        float r = sqrtf(rsq);

        if (r < rmax && r >= rmin)
            {
            // precomputed term
            float value_f = (r - rmin) / delta_r;

            // compute index into the table and read in values
            unsigned int value_i = floor(value_f);
            float2 VF0 = tex1Dfetch(tables_tex, table_value(value_i, cur_bond_type));
            float2 VF1 = tex1Dfetch(tables_tex, table_value(value_i+1, cur_bond_type));
            // unpack the data
            float V0 = VF0.x;
            float V1 = VF1.x;
            float F0 = VF0.y;
            float F1 = VF1.y;

            // compute the linear interpolation coefficient
            float f = value_f - float(value_i);

            // interpolate to get V and F;
            float V = V0 + f * (V1 - V0);
            float F = F0 + f * (F1 - F0);

            // convert to standard variables used by the other pair computes in HOOMD-blue
            float forcemag_divr = 0.0f;
            if (r > 0.0f)
                forcemag_divr = F / r;
            float bond_eng = V;
            // calculate the virial
            float force_div2r = float(0.5) * forcemag_divr;
            virial[0] += dx.x * dx.x * force_div2r; // xx
            virial[1] += dx.x * dx.y * force_div2r; // xy
            virial[2] += dx.x * dx.z * force_div2r; // xz
            virial[3] += dx.y * dx.y * force_div2r; // yy
            virial[4] += dx.y * dx.z * force_div2r; // yz
            virial[5] += dx.z * dx.z * force_div2r; // zz

            // add up the force vector components (FLOPS: 7)
            force.x += dx.x * forcemag_divr;
            force.y += dx.y * forcemag_divr;
            force.z += dx.z * forcemag_divr;
            force.w += bond_eng * 0.5f;
            }
        else
            {
            *d_flags = 1;
            }
        }


    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes);
    d_force[idx] = force;
    for (unsigned int i = 0; i < 6 ; i++)
        d_virial[i*virial_pitch + idx] = virial[i];
    }


/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param box Box dimensions used to implement periodic boundary conditions
    \param blist List of bonds stored on the GPU
    \param pitch Pitch of 2D bond list
    \param n_bonds_list List of numbers of bonds stored on the GPU
    \param n_bond_type number of bond types
    \param d_tables Tables of the potential and force
    \param d_params Parameters for each table associated with a type pair
    \param table_width Number of entries in the table
    \param table_value indexer helper
    \param d_flags flags on the device - a 1 will be written if evaluation
                   of forces failed for any bond
    \param block_size Block size at which to run the kernel

    \note This is just a kernel driver. See gpu_compute_bondtable_forces_kernel for full documentation.
*/
hipError_t gpu_compute_bondtable_forces(float4* d_force,
                                     float* d_virial,
                                     const unsigned int virial_pitch,
                                     const unsigned int N,
                                     const Scalar4 *d_pos,
                                     const BoxDim &box,
                                     const uint2 *blist,
                                     const unsigned int pitch,
                                     const unsigned int *n_bonds_list,
                                     const unsigned int n_bond_type,
                                     const float2 *d_tables,
                                     const float4 *d_params,
                                     const unsigned int table_width,
                                     const Index2D &table_value,
                                     unsigned int *d_flags,
                                     const unsigned int block_size)
    {
    assert(d_params);
    assert(d_tables);
    assert(n_bond_type > 0);
    assert(table_width > 1);


    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);


    // bind the tables texture
    tables_tex.normalized = false;
    tables_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, tables_tex, d_tables, sizeof(float2) * table_value.getNumElements());
    if (error != hipSuccess)
        return error;

    gpu_compute_bondtable_forces_kernel<<< grid, threads, sizeof(float4)*n_bond_type >>>
            (d_force,
             d_virial,
             virial_pitch,
             N,
             d_pos,
             box,
             blist,
             pitch,
             n_bonds_list,
             n_bond_type,
             d_params,
             table_value,
             d_flags);

    return hipSuccess;
    }

// vim:syntax=cpp
