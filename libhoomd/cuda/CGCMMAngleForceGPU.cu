#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: dnlebard

#include "CGCMMAngleForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

// small number. cutoff for igoring the angle as being ill defined.
#define SMALL 0.001f

/*! \file CGCMMAngleForceGPU.cu
    \brief Defines GPU kernel code for calculating the CGCMM angle forces. Used by CGCMMAngleForceComputeGPU.
*/

//! Texture for reading angle parameters
texture<float2, 1, hipReadModeElementType> angle_params_tex;

//! Texture for reading angle CGCMM S-R parameters
texture<float2, 1, hipReadModeElementType> angle_CGCMMsr_tex; // MISSING EPSILON!!! sigma=.x, rcut=.y

//! Texture for reading angle CGCMM Epsilon-pow/pref parameters
texture<float4, 1, hipReadModeElementType> angle_CGCMMepow_tex; // now with EPSILON=.x, pow1=.y, pow2=.z, pref=.w

//! Kernel for caculating CGCMM angle forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param box Box dimensions for periodic boundary condition handling
    \param alist Angle data to use in calculating the forces
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
*/
extern "C" __global__ void gpu_compute_CGCMM_angle_forces_kernel(float4* d_force,
                                                                 float* d_virial,
                                                                 const unsigned int virial_pitch,
                                                                 const unsigned int N,
                                                                 const Scalar4 *d_pos,
                                                                 BoxDim box,
                                                                 const uint4 *alist,
                                                                 const unsigned int pitch,
                                                                 const unsigned int *n_angles_list)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;    

    if (idx >= N)
        return;

    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_angles =n_angles_list[idx];

    // read in the position of our b-particle from the a-b-c triplet. (MEM TRANSFER: 16 bytes)
    float4 idx_postype = d_pos[idx];  // we can be either a, b, or c in the a-b-c triplet
    float3 idx_pos = make_float3(idx_postype.x, idx_postype.y, idx_postype.z);
    float3 a_pos,b_pos,c_pos; // allocate space for the a,b, and c atom in the a-b-c triplet

    // initialize the force to 0
    float4 force_idx = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    float fab[3], fcb[3];
    float fac, eac, vac[6];

    // initialize the virial to 0
    float virial_idx[6];
    for (int i = 0; i < 6; i++)
        virial_idx[i] = 0.0f;

    // loop over all angles
    for (int angle_idx = 0; angle_idx < n_angles; angle_idx++)
        {
        uint4 cur_angle = alist[pitch*angle_idx + idx];

        int cur_angle_x_idx = cur_angle.x;
        int cur_angle_y_idx = cur_angle.y;

        // store the a and c positions to accumlate their forces
        int cur_angle_type = cur_angle.z;
        int cur_angle_abc = cur_angle.w;

        // get the a-particle's position (MEM TRANSFER: 16 bytes)
        float4 x_postype = d_pos[cur_angle_x_idx];
        float3 x_pos = make_float3(x_postype.x, x_postype.y, x_postype.z);
        // get the c-particle's position (MEM TRANSFER: 16 bytes)
        float4 y_postype = d_pos[cur_angle_y_idx];
        float3 y_pos = make_float3(y_postype.x, y_postype.y, y_postype.z);

        if (cur_angle_abc == 0)
            {
            a_pos = idx_pos;
            b_pos = x_pos;
            c_pos = y_pos;
            }
        if (cur_angle_abc == 1)
            {
            b_pos = idx_pos;
            a_pos = x_pos;
            c_pos = y_pos;
            }
        if (cur_angle_abc == 2)
            {
            c_pos = idx_pos;
            a_pos = x_pos;
            b_pos = y_pos;
            }

        // calculate dr for a-b,c-b,and a-c
        Scalar3 dab = a_pos - b_pos;
        Scalar3 dcb = c_pos - b_pos;
        Scalar3 dac = a_pos - c_pos;

        // apply periodic boundary conditions
        dab = box.minImage(dab);
        dcb = box.minImage(dcb);
        dac = box.minImage(dac);

        // get the angle parameters (MEM TRANSFER: 8 bytes)
        float2 params = tex1Dfetch(angle_params_tex, cur_angle_type);
        float K = params.x;
        float t_0 = params.y;

        float rsqab = dot(dab, dab);
        float rab = sqrtf(rsqab);
        float rsqcb = dot(dcb, dcb);;
        float rcb = sqrtf(rsqcb);
        float rsqac = dot(dac, dac);
        float rac = sqrtf(rsqac);

        float c_abbc = dot(dab, dcb);
        c_abbc /= rab*rcb;

        if (c_abbc > 1.0f) c_abbc = 1.0f;
        if (c_abbc < -1.0f) c_abbc = -1.0f;

        float s_abbc = sqrtf(1.0f - c_abbc*c_abbc);
        if (s_abbc < SMALL) s_abbc = SMALL;
        s_abbc = 1.0f/s_abbc;

        //////////////////////////////////////////
        // THIS CODE DOES THE 1-3 LJ repulsions //
        //////////////////////////////////////////////////////////////////////////////
        fac = 0.0f;
        eac = 0.0f;
        for (int i=0; i < 6; i++)
            vac[i] = 0.0f;

        // get the angle E-S-R parameters (MEM TRANSFER: 12 bytes)
        const float2 cgSR = tex1Dfetch(angle_CGCMMsr_tex, cur_angle_type);

        float cgsigma = cgSR.x;
        float cgrcut = cgSR.y;

        if (rac < cgrcut)
            {
            const float4 cgEPOW = tex1Dfetch(angle_CGCMMepow_tex, cur_angle_type);

            // get the angle pow/pref parameters (MEM TRANSFER: 12 bytes)
            float cgeps = cgEPOW.x;
            float cgpow1 = cgEPOW.y;
            float cgpow2 = cgEPOW.z;
            float cgpref = cgEPOW.w;

            float cgratio = cgsigma/rac;
            // INTERESTING NOTE: __powf has weird behavior depending
            // on the inputted parameters.  Try sigma=2.05, versus sigma=0.05
            // in cgcmm_angle_force_test.cc 4 particle test
            fac = cgpref*cgeps / rsqac * (cgpow1*__powf(cgratio,cgpow1) - cgpow2*__powf(cgratio,cgpow2));
            eac = cgeps + cgpref*cgeps * (__powf(cgratio,cgpow1) - __powf(cgratio,cgpow2));

            vac[0] = fac * dac.x*dac.x;
            vac[1] = fac * dac.x*dac.y;
            vac[2] = fac * dac.x*dac.z;
            vac[3] = fac * dac.y*dac.y;
            vac[4] = fac * dac.y*dac.z;
            vac[5] = fac * dac.z*dac.z;
            }
        //////////////////////////////////////////////////////////////////////////////

        // actually calculate the force
        float dth = acosf(c_abbc) - t_0;
        float tk = K*dth;

        float a = -1.0f * tk * s_abbc;
        float a11 = a*c_abbc/rsqab;
        float a12 = -a / (rab*rcb);
        float a22 = a*c_abbc / rsqcb;

        fab[0] = a11*dab.x + a12*dcb.x;
        fab[1] = a11*dab.y + a12*dcb.y;
        fab[2] = a11*dab.z + a12*dcb.z;

        fcb[0] = a22*dcb.x + a12*dab.x;
        fcb[1] = a22*dcb.y + a12*dab.y;
        fcb[2] = a22*dcb.z + a12*dab.z;

        // compute 1/3 of the energy, 1/3 for each atom in the angle
        float angle_eng = (0.5f*tk*dth + eac)*float(1.0f/3.0f);

        float angle_virial[6];
        angle_virial[0] = (1.f/3.f) * ( dab.x*fab[0] + dcb.x*fcb[0] );
        angle_virial[1] = (1.f/3.f) * ( dab.y*fab[0] + dcb.y*fcb[0] );
        angle_virial[2] = (1.f/3.f) * ( dab.z*fab[0] + dcb.z*fcb[0] );
        angle_virial[3] = (1.f/3.f) * ( dab.y*fab[1] + dcb.y*fcb[1] );
        angle_virial[4] = (1.f/3.f) * ( dab.z*fab[1] + dcb.z*fcb[1] );
        angle_virial[5] = (1.f/3.f) * ( dab.z*fab[2] + dcb.z*fcb[2] );

        for (int i = 0; i < 6; i++)
            angle_virial[i] += (1.f/3.f)*vac[i];

        if (cur_angle_abc == 0)
            {
            force_idx.x += fab[0] + fac*dac.x;
            force_idx.y += fab[1] + fac*dac.y;
            force_idx.z += fab[2] + fac*dac.z;
            }
        if (cur_angle_abc == 1)
            {
            force_idx.x -= fab[0] + fcb[0];
            force_idx.y -= fab[1] + fcb[1];
            force_idx.z -= fab[2] + fcb[2];
            }
        if (cur_angle_abc == 2)
            {
            force_idx.x += fcb[0] - fac*dac.x;
            force_idx.y += fcb[1] - fac*dac.y;
            force_idx.z += fcb[2] - fac*dac.z;
            }

        force_idx.w += angle_eng;
        for (int i = 0; i < 6; i++)
            virial_idx[i] += angle_virial[i];
        }

    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    d_force[idx] = force_idx;
    for (int i = 0; i < 6; i++)
        d_virial[i*virial_pitch+idx] = virial_idx[i];
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param d_pos particle positions on the device
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param atable List of angles stored on the GPU
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
    \param d_params K and t_0 params packed as float2 variables
    \param d_CGCMMsr sigma, and rcut packed as a float2
    \param d_CGCMMepow epsilon, pow1, pow2, and prefactor packed as a float4
    \param n_angle_types Number of angle types in d_params
    \param block_size Block size to use when performing calculations

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one float2 element per angle type. The x component contains K the spring constant
    and the y component contains t_0 the equilibrium angle.
*/
hipError_t gpu_compute_CGCMM_angle_forces(float4* d_force,
                                           float* d_virial,
                                           const unsigned int virial_pitch,
                                           const unsigned int N,
                                           const Scalar4 *d_pos,
                                           const BoxDim& box,
                                           const uint4 *atable,
                                           const unsigned int pitch,
                                           const unsigned int *n_angles_list,
                                           float2 *d_params,
                                           float2 *d_CGCMMsr,
                                           float4 *d_CGCMMepow,
                                           unsigned int n_angle_types,
                                           int block_size)
    {
    assert(d_params);
    assert(d_CGCMMsr);
    assert(d_CGCMMepow);


    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // bind the textures
    hipError_t error = hipBindTexture(0, angle_params_tex, d_params, sizeof(float2) * n_angle_types);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, angle_CGCMMsr_tex, d_CGCMMsr, sizeof(float2) * n_angle_types);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, angle_CGCMMepow_tex, d_CGCMMepow, sizeof(float4) * n_angle_types);
    if (error != hipSuccess)
        return error;

    // run the kernel
    gpu_compute_CGCMM_angle_forces_kernel<<< grid, threads>>>(d_force, d_virial, virial_pitch, N, d_pos, box, atable, pitch, n_angles_list);

    return hipSuccess;
    }

