#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


// Maintainer: morozov

/**
powered by:
Moscow group.
*/

#include "EAMForceGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file EAMForceGPU.cu
    \brief Defines GPU kernel code for calculating the eam forces. Used by EAMForceComputeGPU.
*/

//!< Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;
//! Texture for reading electron density
texture<float, 1, hipReadModeElementType> electronDensity_tex;
//! Texture for reading EAM pair potential
texture<float2, 1, hipReadModeElementType> pairPotential_tex;
//! Texture for reading the embedding function
texture<float, 1, hipReadModeElementType> embeddingFunction_tex;
//! Texture for reading the derivative of the electron density
texture<float, 1, hipReadModeElementType> derivativeElectronDensity_tex;
//! Texture for reading the derivative of the embedding function
texture<float, 1, hipReadModeElementType> derivativeEmbeddingFunction_tex;
//! Texture for reading the derivative of the atom embedding function
texture<float, 1, hipReadModeElementType> atomDerivativeEmbeddingFunction_tex;

//! Storage space for EAM parameters on the GPU
__constant__ EAMTexInterData eam_data_ti;

//! Kernel for computing EAM forces on the GPU
extern "C" __global__ void gpu_compute_eam_tex_inter_forces_kernel(
    float4* d_force,
    float* d_virial,
    const unsigned int virial_pitch,
    const unsigned int N,
    const Scalar4 *d_pos,
    BoxDim box,
    const unsigned int *d_n_neigh,
    const unsigned int *d_nlist,
    const Index2D nli,
    float* atomDerivativeEmbeddingFunction)
    {
    // start by identifying which particle we are to handle
    volatile int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the list (MEM_TRANSFER: 4 bytes)
    int n_neigh = d_n_neigh[idx];

    // read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
    // (MEM TRANSFER: 16 bytes)
    float4 postype = tex1Dfetch(pdata_pos_tex, idx);
    float3 pos = make_float3(postype.x, postype.y, postype.z);

    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

    // prefetch neighbor index
    int cur_neigh = 0;
    int next_neigh = d_nlist[nli(idx, 0)];
    int typei  = __float_as_int(postype.w);
    // loop over neighbors

    float atomElectronDensity  = 0.0f;
    int nr = eam_data_ti.nr;
    int ntypes = eam_data_ti.ntypes;
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
        {
        // read the current neighbor index (MEM TRANSFER: 4 bytes)
        // prefetch the next value and set the current one
        cur_neigh = next_neigh;
        next_neigh = d_nlist[nli(idx, neigh_idx+1)];

        // get the neighbor's position (MEM TRANSFER: 16 bytes)
        float4 neigh_postype = tex1Dfetch(pdata_pos_tex, cur_neigh);
        float3 neigh_pos = make_float3(neigh_postype.x, neigh_postype.y, neigh_postype.z);

        // calculate dr (with periodic boundary conditions) (FLOPS: 3)
        float3 dx = pos - neigh_pos;
        int typej  = __float_as_int(neigh_postype.w);
        // apply periodic boundary conditions: (FLOPS 12)
        dx = box.minImage(dx);

        // calculate r squard (FLOPS: 5)
        float rsq = dot(dx, dx);;
        if (rsq < eam_data_ti.r_cutsq)
            {
            float position_float = sqrtf(rsq) * eam_data_ti.rdr;
            atomElectronDensity += tex1D(electronDensity_tex, position_float + nr * (typei * ntypes + typej) + 0.5f ); //electronDensity[r_index + eam_data_ti.nr * typej] + derivativeElectronDensity[r_index + eam_data_ti.nr * typej] * position * eam_data_ti.dr;
            }
        }

    float position = atomElectronDensity * eam_data_ti.rdrho;
    /*unsigned int r_index = (unsigned int)position;
    position -= (float)r_index;*/
    atomDerivativeEmbeddingFunction[idx] = tex1D(derivativeEmbeddingFunction_tex, position + typei * eam_data_ti.nrho + 0.5f);//derivativeEmbeddingFunction[r_index + typei * eam_data_ti.nrho];

    force.w += tex1D(embeddingFunction_tex, position + typei * eam_data_ti.nrho + 0.5f);//embeddingFunction[r_index + typei * eam_data_ti.nrho] + derivativeEmbeddingFunction[r_index + typei * eam_data_ti.nrho] * position * eam_data_ti.drho;
    d_force[idx] = force;
    }

//! Second stage kernel for computing EAM forces on the GPU
extern "C" __global__ void gpu_compute_eam_tex_inter_forces_kernel_2(
    float4* d_force,
    float* d_virial,
    const unsigned int virial_pitch,
    const unsigned int N,
    const Scalar4 *d_pos,
    BoxDim box,
    const unsigned int *d_n_neigh,
    const unsigned int *d_nlist,
    const Index2D nli,
    float* atomDerivativeEmbeddingFunction)
    {
    // start by identifying which particle we are to handle
    volatile  int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // loadj in the length of the list (MEM_TRANSFER: 4 bytes)
    int n_neigh = d_n_neigh[idx];

    // read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
    // (MEM TRANSFER: 16 bytes)
    float4 postype = tex1Dfetch(pdata_pos_tex, idx);
    float3 pos = make_float3(postype.x, postype.y, postype.z);
    int typei = __float_as_int(postype.w);
    // prefetch neighbor index
    float position;
    int cur_neigh = 0;
    int next_neigh = d_nlist[nli(idx, 0)];
    //float4 force = force_data.force[idx];
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    //force.w = force_data.force[idx].w;
    float fxi = 0.0f;
    float fyi = 0.0f;
    float fzi = 0.0f;
    float m_pe = 0.0f;
    float pairForce = 0.0f;
    float virial[6];
    for (int i = 0; i < 6; i++)
        virial[i] = 0.0f;

    force.w = d_force[idx].w;
    int nr = eam_data_ti.nr;
    int ntypes = eam_data_ti.ntypes;
    float adef = atomDerivativeEmbeddingFunction[idx];
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
        {
        cur_neigh = next_neigh;
        next_neigh = d_nlist[nli(idx, neigh_idx+1)];

        // get the neighbor's position (MEM TRANSFER: 16 bytes)
        float4 neigh_postype = tex1Dfetch(pdata_pos_tex,cur_neigh);
        float3 neigh_pos = make_float3(neigh_postype.x, neigh_postype.y, neigh_postype.z);

        // calculate dr (with periodic boundary conditions) (FLOPS: 3)
        float3 dx = pos - neigh_pos;
        int typej = __float_as_int(neigh_postype.w);
        // apply periodic boundary conditions: (FLOPS 12)
        dx = box.minImage(dx);

        // calculate r squard (FLOPS: 5)
        float rsq = dot(dx, dx);

        if (rsq > eam_data_ti.r_cutsq) continue;

        float inverseR = rsqrtf(rsq);
        float r = 1.0f / inverseR;
        position = r * eam_data_ti.rdr;
        int shift = (typei>=typej)?(int)((2 * ntypes - typej -1)*typej/2 + typei) * nr:(int)((2 * ntypes - typei -1)*typei/2 + typej) * nr;
        float2 pair_potential = tex1D(pairPotential_tex, position + shift + 0.5f);
        float pair_eng =  pair_potential.x * inverseR;

        float derivativePhi = (pair_potential.y - pair_eng) * inverseR;

        float derivativeRhoI = tex1D(derivativeElectronDensity_tex, position + typei * eam_data_ti.nr + 0.5f);

        float derivativeRhoJ = tex1D(derivativeElectronDensity_tex, position + typej * eam_data_ti.nr + 0.5f);

        float fullDerivativePhi = adef * derivativeRhoJ +
                atomDerivativeEmbeddingFunction[cur_neigh] * derivativeRhoI + derivativePhi;
        pairForce = - fullDerivativePhi * inverseR;
        float pairForceover2 = 0.5f *pairForce;
        virial[0] += dx.x * dx.x *pairForceover2;
        virial[1] += dx.x * dx.y *pairForceover2;
        virial[2] += dx.x * dx.z *pairForceover2;
        virial[3] += dx.y * dx.y *pairForceover2;
        virial[4] += dx.y * dx.z *pairForceover2;
        virial[5] += dx.z * dx.z *pairForceover2;

        fxi += dx.x * pairForce;
        fyi += dx.y * pairForce;
        fzi += dx.z * pairForce;
        m_pe += pair_eng * 0.5f;
        }
    
    force.x = fxi;
    force.y = fyi;
    force.z = fzi;
    force.w += m_pe;
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    d_force[idx] = force;
    for (int i = 0; i < 6; i++)
        d_virial[i*virial_pitch+idx] = virial[i];
    }

hipError_t gpu_compute_eam_tex_inter_forces(
    float4* d_force,
    float* d_virial,
    const unsigned int virial_pitch,
    const unsigned int N,
    const Scalar4 *d_pos,
    const BoxDim& box,
    const unsigned int *d_n_neigh,
    const unsigned int *d_nlist,
    const Index2D& nli,
    const EAMtex& eam_tex,
    const EAMTexInterArrays& eam_arrays,
    const EAMTexInterData& eam_data)
    {
    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)eam_data.block_size), 1, 1);
    dim3 threads(eam_data.block_size, 1, 1);

    // bind the texture
    pdata_pos_tex.normalized = false;
    pdata_pos_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, pdata_pos_tex, d_pos, sizeof(float4)*N);
    if (error != hipSuccess)
        return error;

    electronDensity_tex.normalized = false;
    electronDensity_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(electronDensity_tex, eam_tex.electronDensity);
    if (error != hipSuccess)
        return error;

    pairPotential_tex.normalized = false;
    pairPotential_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(pairPotential_tex, eam_tex.pairPotential);
    if (error != hipSuccess)
        return error;

    embeddingFunction_tex.normalized = false;
    embeddingFunction_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(embeddingFunction_tex, eam_tex.embeddingFunction);
    if (error != hipSuccess)
        return error;

    derivativeElectronDensity_tex.normalized = false;
    derivativeElectronDensity_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(derivativeElectronDensity_tex, eam_tex.derivativeElectronDensity);
    if (error != hipSuccess)
        return error;

    derivativeEmbeddingFunction_tex.normalized = false;
    derivativeEmbeddingFunction_tex.filterMode = hipFilterModeLinear ;
    error = hipBindTextureToArray(derivativeEmbeddingFunction_tex, eam_tex.derivativeEmbeddingFunction);
    if (error != hipSuccess)
        return error;
    // run the kernel
    hipMemcpyToSymbol(HIP_SYMBOL("eam_data_ti"), &eam_data, sizeof(EAMTexInterData));

    gpu_compute_eam_tex_inter_forces_kernel<<< grid, threads>>>(d_force,
                                                                d_virial,
                                                                virial_pitch,
                                                                N,
                                                                d_pos,
                                                                box,
                                                                d_n_neigh,
                                                                d_nlist,
                                                                nli,
                                                                eam_arrays.atomDerivativeEmbeddingFunction);

    gpu_compute_eam_tex_inter_forces_kernel_2<<< grid, threads>>>(d_force,
                                                                  d_virial,
                                                                  virial_pitch,
                                                                  N,
                                                                  d_pos,
                                                                  box,
                                                                  d_n_neigh,
                                                                  d_nlist,
                                                                  nli,
                                                                  eam_arrays.atomDerivativeEmbeddingFunction);

    return hipSuccess;
    }

// vim:syntax=cpp

