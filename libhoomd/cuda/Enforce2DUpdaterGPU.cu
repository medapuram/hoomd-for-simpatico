#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "Enforce2DUpdaterGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

#include <stdio.h>

/*! \file Enforce2DUpdaterGPU.cu
    \brief Defines GPU kernel code for constraining systems to a 2D plane on 
    the GPU. Used by Enforce2DUpdaterGPU.
*/

//! Constrains partcles to the xy plane on the GPU
/*! \param N number of particles in system
    \param d_vel Particle velocities to constrain to xy plane
    \param d_accel Particle accelerations to constrain to xy plane
*/
extern "C" __global__ 
void gpu_enforce2d_kernel(const unsigned int N,
                          Scalar4 *d_vel,
                          Scalar3 *d_accel)
    {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N)
        {        
        // read the particle's velocity and acceleration (MEM TRANSFER: 32 bytes)
        Scalar4 vel = d_vel[idx];
        Scalar3 accel = d_accel[idx];
                
        // zero the z-velocity and z-acceleration(FLOPS: ?)
        vel.z = 0.0f;
        accel.z = 0.0f;
                
        // write out the results (MEM_TRANSFER: 32 bytes)
        d_vel[idx] = vel;
        d_accel[idx] = accel;
        }
    }

/*! \param N number of particles in system
    \param d_vel Particle velocities to constrain to xy plane
    \param d_accel Particle accelerations to constrain to xy plane
*/
hipError_t gpu_enforce2d(const unsigned int N,
                          Scalar4 *d_vel,
                          Scalar3 *d_accel)
    {
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (N/block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);
            
    // run the kernel
    gpu_enforce2d_kernel<<< grid, threads >>>(N, d_vel, d_accel);
    
    return hipSuccess;
    }

