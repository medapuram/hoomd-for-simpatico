#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "Integrator.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file Integrator.cu
    \brief Defines methods and data structures used by the Integrator class on the GPU
*/

//! helper to add a given force/virial pointer pair
template< unsigned int compute_virial >
__device__ void add_force_total(float4& net_force, float *net_virial, float4& net_torque, float4* d_f, float* d_v, const unsigned int virial_pitch, float4* d_t, int idx)
    {
    if (d_f != NULL && d_v != NULL && d_t != NULL)
        {
        float4 f = d_f[idx];
        float4 t = d_t[idx];
        
        net_force.x += f.x;
        net_force.y += f.y;
        net_force.z += f.z;
        net_force.w += f.w;

        if (compute_virial)
            {
            for (int i=0; i < 6; i++)
                net_virial[i] += d_v[i*virial_pitch+idx];
            }
        
        net_torque.x += t.x;
        net_torque.y += t.y;
        net_torque.z += t.z;
        net_torque.w += t.w;
        }
    }

//! Kernel for summing forces on the GPU
/*! The speicified forces and virials are summed for every particle into \a d_net_force and \a d_net_virial

    \param d_net_force Output device array to hold the computed net force
    \param d_net_virial Output device array to hold the computed net virial
    \param net_virial_pitch The pitch of the 2D net_virial array
    \param d_net_torque Output device array to hold the computed net torque
    \param force_list List of pointers to force data to sum
    \param nparticles Number of particles in the arrays
    \param clear When true, initializes the sums to 0 before adding. When false, reads in the current \a d_net_force
           and \a d_net_virial and adds to that

    \tparam compute_virial When set to 0, the virial sum is not computed
*/
template< unsigned int compute_virial >
__global__ void gpu_integrator_sum_net_force_kernel(float4 *d_net_force,
                                                    float *d_net_virial,
                                                    const unsigned int net_virial_pitch,
                                                    float4 *d_net_torque,
                                                    const gpu_force_list force_list,
                                                    unsigned int nparticles,
                                                    bool clear)
    {
    // calculate the index we will be handling
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx < nparticles)
        {
        // set the initial net_force and net_virial to sum into
        float4 net_force;
        float net_virial[6];
        float4 net_torque;
        if (clear)
            {
            net_force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            if (compute_virial)
                {
                for (int i=0; i<6; i++)
                    net_virial[i] = 0.0f;
                }
            net_torque = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
            }
        else
            {
            // if clear is false, intialize to the current d_net_force and d_net_virial
            net_force = d_net_force[idx];
            if (compute_virial)
                {
                for (int i=0; i<6; i++)
                    net_virial[i] = d_net_virial[i*net_virial_pitch+idx];
                }
            net_torque = d_net_torque[idx];
            }
        
        // sum up the totals
        add_force_total<compute_virial>(net_force, net_virial, net_torque, force_list.f0, force_list.v0, force_list.vpitch0, force_list.t0, idx);
        add_force_total<compute_virial>(net_force, net_virial, net_torque, force_list.f1, force_list.v1, force_list.vpitch1, force_list.t1, idx);
        add_force_total<compute_virial>(net_force, net_virial, net_torque, force_list.f2, force_list.v2, force_list.vpitch2, force_list.t2, idx);
        add_force_total<compute_virial>(net_force, net_virial, net_torque, force_list.f3, force_list.v3, force_list.vpitch3, force_list.t3, idx);
        add_force_total<compute_virial>(net_force, net_virial, net_torque, force_list.f4, force_list.v4, force_list.vpitch4, force_list.t4, idx);
        add_force_total<compute_virial>(net_force, net_virial, net_torque, force_list.f5, force_list.v5, force_list.vpitch5, force_list.t5, idx);
        
        // write out the final result
        d_net_force[idx] = net_force;
        if (compute_virial)
            {
            for (int i=0; i < 6; i++)
                d_net_virial[i*net_virial_pitch+idx] = net_virial[i];
            }
        d_net_torque[idx] = net_torque;
        }
    }

hipError_t gpu_integrator_sum_net_force(float4 *d_net_force,
                                         float *d_net_virial,
                                         const unsigned int net_virial_pitch,
                                         float4 *d_net_torque,
                                         const gpu_force_list& force_list,
                                         unsigned int nparticles,
                                         bool clear,
                                         bool compute_virial)
    {
    // sanity check
    assert(d_net_force);
    assert(d_net_virial);
    assert(d_net_torque);
    
    const int block_size = 256;
    
    if (compute_virial)
        {
        gpu_integrator_sum_net_force_kernel<1><<< nparticles/block_size+1, block_size >>>(d_net_force,
                                                                                          d_net_virial,
                                                                                          net_virial_pitch,
                                                                                          d_net_torque,
                                                                                          force_list,
                                                                                          nparticles,
                                                                                          clear);
        }
    else
        {
        gpu_integrator_sum_net_force_kernel<0><<< nparticles/block_size+1, block_size >>>(d_net_force,
                                                                                          d_net_virial,
                                                                                          net_virial_pitch,
                                                                                          d_net_torque,
                                                                                          force_list,
                                                                                          nparticles,
                                                                                          clear);
        }
    
    return hipSuccess;
    }

