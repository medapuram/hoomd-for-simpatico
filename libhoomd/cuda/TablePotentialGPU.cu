#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "TablePotentialGPU.cuh"

#include "Index1D.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TablePotentialGPU.cu
    \brief Defines GPU kernel code for calculating the table pair forces. Used by TablePotentialGPU.
*/

//! Texture for reading particle positions
texture<float4, 1, hipReadModeElementType> pdata_pos_tex;

//! Texture for reading table values
texture<float2, 1, hipReadModeElementType> tables_tex;

/*!  This kernel is called to calculate the table pair forces on all N particles

    \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch Pitch of 2D virial array
    \param N number of particles in system
    \param d_pos device array of particle positions
    \param box Box dimensions used to implement periodic boundary conditions
    \param d_n_neigh Device memory array listing the number of neighbors for each particle
    \param d_nlist Device memory array containing the neighbor list contents
    \param nli Indexer for indexing \a d_nlist
    \param d_params Parameters for each table associated with a type pair
    \param ntypes Number of particle types in the system
    \param table_width Number of points in each table

    See TablePotential for information on the memory layout.

    \b Details:
    * Table entries are read from tables_tex. Note that currently this is bound to a 1D memory region. Performance tests
      at a later date may result in this changing.
*/
__global__ void gpu_compute_table_forces_kernel(float4* d_force,
                                                float* d_virial,
                                                const unsigned virial_pitch,
                                                const unsigned int N,
                                                const Scalar4 *d_pos,
                                                const BoxDim box,
                                                const unsigned int *d_n_neigh,
                                                const unsigned int *d_nlist,
                                                const Index2D nli,
                                                const float4 *d_params,
                                                const unsigned int ntypes,
                                                const unsigned int table_width)
    {
    // index calculation helpers
    Index2DUpperTriangular table_index(ntypes);
    Index2D table_value(table_width);

    // read in params for easy and fast access in the kernel
    extern __shared__ float4 s_params[];
    for (unsigned int cur_offset = 0; cur_offset < table_index.getNumElements(); cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < table_index.getNumElements())
            s_params[cur_offset + threadIdx.x] = d_params[cur_offset + threadIdx.x];
        }
    __syncthreads();

    // start by identifying which particle we are to handle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the list
    unsigned int n_neigh = d_n_neigh[idx];

    // read in the position of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
    Scalar4 postype = tex1Dfetch(pdata_pos_tex, idx);
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int typei = __float_as_int(postype.w);

    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float virialxx = 0.0f;
    float virialxy = 0.0f;
    float virialxz = 0.0f;
    float virialyy = 0.0f;
    float virialyz = 0.0f;
    float virialzz = 0.0f;

    // prefetch neighbor index
    unsigned int cur_neigh = 0;
    unsigned int next_neigh = d_nlist[nli(idx, 0)];

    // loop over neighbors
    // on pre Fermi hardware, there is a bug that causes rare and random ULFs when simply looping over n_neigh
    // the workaround (activated via the template paramter) is to loop over nlist.height and put an if (i < n_neigh)
    // inside the loop
    #if (__CUDA_ARCH__ < 200)
    for (int neigh_idx = 0; neigh_idx < nli.getH(); neigh_idx++)
    #else
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
    #endif
        {
        #if (__CUDA_ARCH__ < 200)
        if (neigh_idx < n_neigh)
        #endif
            {
            // read the current neighbor index
            // prefetch the next value and set the current one
            cur_neigh = next_neigh;
            next_neigh = d_nlist[nli(idx, (neigh_idx+1))];

            // get the neighbor's position
            float4 neigh_postype = tex1Dfetch(pdata_pos_tex, cur_neigh);
            float3 neigh_pos = make_float3(neigh_postype.x, neigh_postype.y, neigh_postype.z);

            // calculate dr (with periodic boundary conditions)
            float3 dx = pos - neigh_pos;

            // apply periodic boundary conditions
            dx = box.minImage(dx);

            // access needed parameters
            unsigned int typej = __float_as_int(neigh_postype.w);
            unsigned int cur_table_index = table_index(typei, typej);
            float4 params = s_params[cur_table_index];
            float rmin = params.x;
            float rmax = params.y;
            float delta_r = params.z;

            // calculate r
            float rsq = dot(dx, dx);
            float r = sqrtf(rsq);

            if (r < rmax && r >= rmin)
                {
                // precomputed term
                float value_f = (r - rmin) / delta_r;

                // compute index into the table and read in values
                unsigned int value_i = floor(value_f);
                float2 VF0 = tex1Dfetch(tables_tex, table_value(value_i, cur_table_index));
                float2 VF1 = tex1Dfetch(tables_tex, table_value(value_i+1, cur_table_index));
                // unpack the data
                float V0 = VF0.x;
                float V1 = VF1.x;
                float F0 = VF0.y;
                float F1 = VF1.y;

                // compute the linear interpolation coefficient
                float f = value_f - float(value_i);

                // interpolate to get V and F;
                float V = V0 + f * (V1 - V0);
                float F = F0 + f * (F1 - F0);

                // convert to standard variables used by the other pair computes in HOOMD-blue
                float forcemag_divr = 0.0f;
                if (r > 0.0f)
                    forcemag_divr = F / r;
                float pair_eng = V;
                // calculate the virial
                float force_div2r = float(0.5) * forcemag_divr;
                virialxx +=  dx.x * dx.x * force_div2r;
                virialxy +=  dx.x * dx.y * force_div2r;
                virialxz +=  dx.x * dx.z * force_div2r;
                virialyy +=  dx.y * dx.y * force_div2r;
                virialyz +=  dx.y * dx.z * force_div2r;
                virialzz +=  dx.z * dx.z * force_div2r;

                // add up the force vector components (FLOPS: 7)
                force.x += dx.x * forcemag_divr;
                force.y += dx.y * forcemag_divr;
                force.z += dx.z * forcemag_divr;
                force.w += pair_eng;
                }
            }
        }

    // potential energy per particle must be halved
    force.w *= 0.5f;
    // now that the force calculation is complete, write out the result
    d_force[idx] = force;
    d_virial[0*virial_pitch+idx] = virialxx;
    d_virial[1*virial_pitch+idx] = virialxy;
    d_virial[2*virial_pitch+idx] = virialxz;
    d_virial[3*virial_pitch+idx] = virialyy;
    d_virial[4*virial_pitch+idx] = virialyz;
    d_virial[5*virial_pitch+idx] = virialzz;
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param n_ghost number of ghost particles
    \param d_pos particle positions on the device
    \param box Box dimensions used to implement periodic boundary conditions
    \param d_n_neigh Device memory array listing the number of neighbors for each particle
    \param d_nlist Device memory array containing the neighbor list contents
    \param nli Indexer for indexing \a d_nlist
    \param d_tables Tables of the potential and force
    \param d_params Parameters for each table associated with a type pair
    \param ntypes Number of particle types in the system
    \param table_width Number of points in each table
    \param block_size Block size at which to run the kernel

    \note This is just a kernel driver. See gpu_compute_table_forces_kernel for full documentation.
*/
hipError_t gpu_compute_table_forces(float4* d_force,
                                     float* d_virial,
                                     const unsigned int virial_pitch,
                                     const unsigned int N,
                                     const unsigned int n_ghost,
                                     const Scalar4 *d_pos,
                                     const BoxDim& box,
                                     const unsigned int *d_n_neigh,
                                     const unsigned int *d_nlist,
                                     const Index2D& nli,
                                     const float2 *d_tables,
                                     const float4 *d_params,
                                     const unsigned int ntypes,
                                     const unsigned int table_width,
                                     const unsigned int block_size)
    {
    assert(d_params);
    assert(d_tables);
    assert(ntypes > 0);
    assert(table_width > 1);

    // index calculation helper
    Index2DUpperTriangular table_index(ntypes);

    // setup the grid to run the kernel
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);

    // bind the pdata position texture
    pdata_pos_tex.normalized = false;
    pdata_pos_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, pdata_pos_tex, d_pos, sizeof(float4) * (N+n_ghost));
    if (error != hipSuccess)
        return error;

    // bind the tables texture
    tables_tex.normalized = false;
    tables_tex.filterMode = hipFilterModePoint;
    error = hipBindTexture(0, tables_tex, d_tables, sizeof(float2) * table_width * table_index.getNumElements());
    if (error != hipSuccess)
        return error;

    gpu_compute_table_forces_kernel<<< grid, threads, sizeof(float4)*table_index.getNumElements() >>>
            (d_force, d_virial, virial_pitch, N, d_pos, box, d_n_neigh, d_nlist, nli, d_params, ntypes, table_width);

    return hipSuccess;
    }

// vim:syntax=cpp

