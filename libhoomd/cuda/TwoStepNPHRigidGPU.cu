#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: ndtrung

#include "QuaternionMath.h"
#include "TwoStepNPTRigidGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TwoStepNPTRigidGPU.cu
    \brief Defines GPU kernel code for NPT integration on the GPU. Used by TwoStepNPTRigidGPU.
*/

// Flag for invalid particle index, identical to the sentinel value NO_INDEX in RigidData.h
#define INVALID_INDEX 0xffffffff

/*! Maclaurine expansion
    \param x Point to take the expansion

*/
__device__ float nph_maclaurin_series(float x)
    {
    float x2, x4;
    x2 = x * x;
    x4 = x2 * x2;
    return (1.0f + (1.0f/6.0f) * x2 + (1.0f/120.0f) * x4 + (1.0f/5040.0f) * x2 * x4 + (1.0f/362880.0f) * x4 * x4);
    }

/*! Kernel to zero virial contribution from particles from rigid bodies
    \param d_virial_rigid Virial contribution from particles in rigid bodies
    \param local_num Number of particles in this card
*/
extern "C" __global__ void gpu_nph_rigid_zero_virial_rigid_kernel(float *d_virial_rigid, 
                                                                 unsigned int local_num)
    {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // particle's index

    if (idx < local_num)
        {
        d_virial_rigid[idx] = 0.0f;
        }

    }

/*! Takes the first half-step forward for rigid bodies in the velocity-verlet NVT integration 
    \param rdata_com Body center of mass
    \param d_rigid_group Body group
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total umber of rigid bodies
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata_dilation Volume scaling factor
    \param npt_rdata_dimension System dimensionality
    \param npt_rdata_new box New box sizes
*/

extern "C" __global__ void gpu_nph_rigid_remap_kernel(float4 *rdata_com,
                                                      unsigned int *d_rigid_group,
                                                      unsigned int n_group_bodies,
                                                      unsigned int n_bodies, 
                                                      BoxDim box,
                                                      float nph_rdata_dilation,
                                                      unsigned int nph_rdata_dimension,
                                                      float4 *nph_rdata_new_box)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= n_group_bodies)
        return;

    unsigned int idx_body = d_rigid_group[group_idx];

    Scalar3 curL = box.getL();
    float3 L;

    // reset box to new size/shape
    L.x = curL.x * nph_rdata_dilation;
    L.y = curL.y * nph_rdata_dilation;
    if (nph_rdata_dimension == 3)
        L.z = curL.z * nph_rdata_dilation;
    
    // copy and setL 
    BoxDim newBox = box;
    newBox.setL(L);

    float4 com = rdata_com[idx_body];
    Scalar3 f = box.makeFraction(make_scalar3(com.x, com.y, com.z));
    Scalar3 pos = newBox.makeCoordinates(f);

    // write out results
    rdata_com[idx_body] = make_float4(pos.x, pos.y, pos.z, 0.0f);

    if (idx_body == 0)
        {
        *(nph_rdata_new_box) = make_float4(L.x, L.y, L.z, 0.0f);
        }
    }


#pragma mark RIGID_STEP_ONE_KERNEL
/*! Takes the first half-step forward for rigid bodies in the velocity-verlet NVT integration 
    \param rdata_com Body center of mass
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_body_image Body image 
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param n_group_bodies Number of rigid bodies in my group
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_bodies Total umber of rigid bodies
    \param npt_rdata_epsilon_dot Barostat velocity
    \param npt_rdata_partial_Ksum_t Body translational kinetic energy 
    \param npt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param npt_rdata_nf_t Translational degrees of freedom
    \param npt_rdata_nf_r Translational degrees of freedom
    \param npt_rdata_dimension System dimesion
    \param box Box dimensions for periodic boundary condition handling
    \param deltaT Timestep 
    
*/

extern "C" __global__ void gpu_nph_rigid_step_one_body_kernel(float4* rdata_com, 
                                                            float4* rdata_vel, 
                                                            float4* rdata_angmom, 
                                                            float4* rdata_angvel,
                                                            float4* rdata_orientation, 
                                                            int3* rdata_body_image, 
                                                            float4* rdata_conjqm,
                                                            float *d_rigid_mass,
                                                            float4 *d_rigid_mi,
                                                            float4 *d_rigid_force,
                                                            float4 *d_rigid_torque,
                                                            unsigned int *d_rigid_group, 
                                                            unsigned int n_group_bodies,  
                                                            unsigned int n_bodies, 
                                                            float npt_rdata_epsilon_dot, 
                                                            float* npt_rdata_partial_Ksum_t, 
                                                            float* npt_rdata_partial_Ksum_r,
                                                            unsigned int npt_rdata_nf_t,
                                                            unsigned int npt_rdata_nf_r,
                                                            unsigned int npt_rdata_dimension, 
                                                            BoxDim box, 
                                                            float deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx >= n_group_bodies)
        return;

    // do velocity verlet update
    // v(t+deltaT/2) = v(t) + (1/2)a*deltaT
    // r(t+deltaT) = r(t) + v(t+deltaT/2)*deltaT
    float body_mass;
    float4 moment_inertia, com, vel, orientation, ex_space, ey_space, ez_space, force, torque, conjqm;
    int3 body_image;
    float4 mbody, tbody, fquat;

    float dt_half = 0.5f * deltaT;
    float onednft, onednfr, tmp, scale_t, scale_r, scale_v, akin_t, akin_r;

    onednft = 1.0f + (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_t+npt_rdata_nf_r);
    onednfr = (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_t+npt_rdata_nf_r);

    tmp = -1.0f * dt_half * onednft * npt_rdata_epsilon_dot;
    scale_t = __expf(tmp);
    tmp = -1.0f * dt_half * onednfr * npt_rdata_epsilon_dot;
    scale_r = __expf(tmp);
    tmp = dt_half * npt_rdata_epsilon_dot;
    scale_v = deltaT * __expf(tmp) * nph_maclaurin_series(tmp);

    unsigned int idx_body = d_rigid_group[group_idx];
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    com = rdata_com[idx_body];
    vel = rdata_vel[idx_body];
    orientation = rdata_orientation[idx_body];
    body_image = rdata_body_image[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    conjqm = rdata_conjqm[idx_body];

    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);

    // update velocity
    float dtfm = dt_half / body_mass;

    float4 vel2;
    vel2.x = vel.x + dtfm * force.x;
    vel2.y = vel.y + dtfm * force.y;
    vel2.z = vel.z + dtfm * force.z;
    vel2.x *= scale_t;
    vel2.y *= scale_t;
    vel2.z *= scale_t;
    vel2.w = vel.w;

    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;

    // update position
    float3 pos2;
    pos2.x = com.x + vel2.x * scale_v;
    pos2.y = com.y + vel2.y * scale_v;
    pos2.z = com.z + vel2.z * scale_v;

    // time to fix the periodic boundary conditions
    box.wrap(pos2, body_image);

    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);

    float4 conjqm2;
    conjqm2.x = conjqm.x + deltaT * fquat.x;
    conjqm2.y = conjqm.y + deltaT * fquat.y;
    conjqm2.z = conjqm.z + deltaT * fquat.z;
    conjqm2.w = conjqm.w + deltaT * fquat.w;

    conjqm2.x *= scale_r;
    conjqm2.y *= scale_r;
    conjqm2.z *= scale_r;
    conjqm2.w *= scale_r;

    // use no_squish rotate to update p and q
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(1, conjqm2, orientation, moment_inertia, deltaT);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);

    // update the exyz_space
    // transform p back to angmom
    // update angular velocity
    float4 angmom2;
    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);
    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);

    angmom2.x *= 0.5f;
    angmom2.y *= 0.5f;
    angmom2.z *= 0.5f;

    float4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);

    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;

    // write out the results (MEM_TRANSFER: ? bytes)
    rdata_com[idx_body] = make_float4(pos2.x, pos2.y, pos2.z, com.w);
    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_orientation[idx_body] = orientation;
    rdata_body_image[idx_body] = body_image;
    rdata_conjqm[idx_body] = conjqm2;

    npt_rdata_partial_Ksum_t[group_idx] = akin_t;
    npt_rdata_partial_Ksum_r[group_idx] = akin_r;
    }

/*! \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata Thermostat/barostat data
    \param deltaT Amount of real time to step forward in one time step
    
*/
hipError_t gpu_nph_rigid_step_one(const gpu_rigid_data_arrays& rigid_data,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   float4 *d_net_force,
                                   const BoxDim& box, 
                                   const gpu_npt_rigid_data& npt_rdata,
                                   float deltaT)
    {
    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;
    
    // setup the grid to run the kernel for rigid bodies
    int block_size = 64;
    int n_blocks = n_group_bodies / block_size + 1;
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);
    gpu_nph_rigid_step_one_body_kernel<<< body_grid, body_threads  >>>(rigid_data.com, 
                                                                       rigid_data.vel, 
                                                                       rigid_data.angmom, 
                                                                       rigid_data.angvel,
                                                                       rigid_data.orientation, 
                                                                       rigid_data.body_image, 
                                                                       rigid_data.conjqm,
                                                                       rigid_data.body_mass,
                                                                       rigid_data.moment_inertia,
                                                                       rigid_data.force,
                                                                       rigid_data.torque,
                                                                       rigid_data.body_indices,
                                                                       n_group_bodies,
                                                                       n_bodies, 
                                                                       npt_rdata.epsilon_dot, 
                                                                       npt_rdata.partial_Ksum_t,
                                                                       npt_rdata.partial_Ksum_r,
                                                                       npt_rdata.nf_t,
                                                                       npt_rdata.nf_r,
                                                                       npt_rdata.dimension,
                                                                       box, 
                                                                       deltaT);
       
    gpu_nph_rigid_remap_kernel<<< body_grid, body_threads >>>(rigid_data.com,
                                                              rigid_data.body_indices,
                                                              n_group_bodies,
                                                              n_bodies,
                                                              box, 
                                                              npt_rdata.dilation,
                                                              npt_rdata.dimension,
                                                              npt_rdata.new_box);

                                                                    
    return hipSuccess;
    }

#pragma mark RIGID_STEP_TWO_KERNEL
//! Takes the 2nd 1/2 step forward in the velocity-verlet NPH integration scheme
/*!  
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total number of rigid bodies
    \param npt_rdata_eta_dot_t0 Thermostat translational part 
    \param npt_rdata_eta_dot_r0 Thermostat rotational part
    \param npt_rdata_epsilon_dot Barostat velocity
    \param npt_rdata_partial_Ksum_t Body translational kinetic energy 
    \param npt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param npt_rdata_nf_t Translational degrees of freedom
    \param npt_rdata_nf_r Translational degrees of freedom
    \param npt_rdata_dimension System dimesion
    \param deltaT Timestep 
    \param box Box dimensions for periodic boundary condition handling
*/

extern "C" __global__ void gpu_nph_rigid_step_two_body_kernel(float4* rdata_vel, 
                                                              float4* rdata_angmom, 
                                                              float4* rdata_angvel,
                                                              float4* rdata_orientation,
                                                              float4* rdata_conjqm,
                                                              float *d_rigid_mass,
                                                              float4 *d_rigid_mi,
                                                              float4 *d_rigid_force,
                                                              float4 *d_rigid_torque,
                                                              unsigned int *d_rigid_group,
                                                              unsigned int n_group_bodies,
                                                              unsigned int n_bodies, 
                                                              float npt_rdata_epsilon_dot, 
                                                              float* npt_rdata_partial_Ksum_t,
                                                              float* npt_rdata_partial_Ksum_r,
                                                              unsigned int npt_rdata_nf_t,
                                                              unsigned int npt_rdata_nf_r,
                                                              unsigned int npt_rdata_dimension,
                                                              BoxDim box, 
                                                              float deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (group_idx >= n_group_bodies)
        return;
        
    float body_mass;
    float4 moment_inertia, vel, ex_space, ey_space, ez_space, orientation, conjqm;
    float4 force, torque;
    float4 mbody, tbody, fquat;
    
    float dt_half = 0.5f * deltaT;
    float onednft, onednfr, tmp, scale_t, scale_r, akin_t, akin_r;
    
    onednft = 1.0f + (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_t+npt_rdata_nf_r);
    onednfr = (float) (npt_rdata_dimension) / (float) (npt_rdata_nf_t+npt_rdata_nf_r);

    tmp = -1.0f * dt_half * onednft * npt_rdata_epsilon_dot;
    scale_t = exp(tmp);
    tmp = -1.0f * dt_half * onednfr * npt_rdata_epsilon_dot;
    scale_r = exp(tmp);
    
    unsigned int idx_body = d_rigid_group[group_idx];

    // Update body velocity and angmom
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    vel = rdata_vel[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    orientation = rdata_orientation[idx_body];
    conjqm = rdata_conjqm[idx_body];
    
    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);
    
    float dtfm = dt_half / body_mass;
    
    // update the velocity
    float4 vel2;
    vel2.x = scale_t * vel.x + dtfm * force.x;
    vel2.y = scale_t * vel.y + dtfm * force.y;
    vel2.z = scale_t * vel.z + dtfm * force.z;
    vel2.w = 0.0f;
    
    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;
    
    // update angular momentum
    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);
    
    float4  conjqm2, angmom2;
    conjqm2.x = scale_r * conjqm.x + deltaT * fquat.x;
    conjqm2.y = scale_r * conjqm.y + deltaT * fquat.y;
    conjqm2.z = scale_r * conjqm.z + deltaT * fquat.z;
    conjqm2.w = scale_r * conjqm.w + deltaT * fquat.w;
    
    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);
    
    angmom2.x *= 0.5f;
    angmom2.y *= 0.5f;
    angmom2.z *= 0.5f;
    angmom2.w = 0.0f;
    
    // update angular velocity
    float4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);
    
    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;
    
    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_conjqm[idx_body] = conjqm2;
    
    npt_rdata_partial_Ksum_t[group_idx] = akin_t;
    npt_rdata_partial_Ksum_r[group_idx] = akin_r;
    }

/*! \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param d_net_virial Particle net virial
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata Thermostat/barostat data
    \param deltaT Amount of real time to step forward in one time step
    
*/
hipError_t gpu_nph_rigid_step_two( const gpu_rigid_data_arrays& rigid_data,
                                    unsigned int *d_group_members,
                                    unsigned int group_size,
                                    float4 *d_net_force,
                                    float *d_net_virial,
                                    const BoxDim& box, 
                                    const gpu_npt_rigid_data& npt_rdata,
                                    float deltaT)
    {
    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;
                                                                                                                                                            
    unsigned int block_size = 64;
    unsigned int n_blocks = n_group_bodies / block_size + 1;                                
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);                                                 
    gpu_nph_rigid_step_two_body_kernel<<< body_grid, body_threads >>>(rigid_data.vel, 
                                                                    rigid_data.angmom, 
                                                                    rigid_data.angvel,
                                                                    rigid_data.orientation,
                                                                    rigid_data.conjqm,
                                                                    rigid_data.body_mass,
                                                                    rigid_data.moment_inertia,
                                                                    rigid_data.force,
                                                                    rigid_data.torque,
                                                                    rigid_data.body_indices,
                                                                    n_group_bodies,
                                                                    n_bodies, 
                                                                    npt_rdata.epsilon_dot, 
                                                                    npt_rdata.partial_Ksum_t,
                                                                    npt_rdata.partial_Ksum_r,
                                                                    npt_rdata.nf_t,
                                                                    npt_rdata.nf_r,
                                                                    npt_rdata.dimension, 
                                                                    box, 
                                                                    deltaT);

                                                                                                                                             
    return hipSuccess;
    }

#pragma mark RIGID_KINETIC_ENERGY_REDUCTION

//! Shared memory for kinetic energy reduction
extern __shared__ float nph_rigid_sdata[];

/*! Summing the kinetic energy of rigid bodies
    \param npt_rdata Thermostat data for rigid bodies 
    
*/
extern "C" __global__ void gpu_nph_rigid_reduce_ksum_kernel(float* npt_rdata_partial_Ksum_t,
                                                            float* npt_rdata_partial_Ksum_r,
                                                            float* npt_rdata_Ksum_t,
                                                            float* npt_rdata_Ksum_r,
                                                            unsigned int n_bodies)
    {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    float* body_ke_t = nph_rigid_sdata;
    float* body_ke_r = &nph_rigid_sdata[blockDim.x];
    
    float Ksum_t = 0.0f, Ksum_r=0.0f;
    
    // sum up the values in the partial sum via a sliding window
    for (int start = 0; start < n_bodies; start += blockDim.x)
        {
        if (start + threadIdx.x < n_bodies)
            {
            body_ke_t[threadIdx.x] = npt_rdata_partial_Ksum_t[start + threadIdx.x];
            body_ke_r[threadIdx.x] = npt_rdata_partial_Ksum_r[start + threadIdx.x];
            }
        else
            {
            body_ke_t[threadIdx.x] = 0.0f;
            body_ke_r[threadIdx.x] = 0.0f;
            }
        __syncthreads();
        
        // reduce the sum within a block
        int offset = blockDim.x >> 1;
        while (offset > 0)
            {
            if (threadIdx.x < offset)
                {
                body_ke_t[threadIdx.x] += body_ke_t[threadIdx.x + offset];
                body_ke_r[threadIdx.x] += body_ke_r[threadIdx.x + offset];
                }
            offset >>= 1;
            __syncthreads();
            }
            
        // everybody sums up Ksum
        Ksum_t += body_ke_t[0];
        Ksum_r += body_ke_r[0];
        }
        
    __syncthreads();
    
    
    if (global_idx == 0)
        {
        *npt_rdata_Ksum_t = Ksum_t;
        *npt_rdata_Ksum_r = Ksum_r;
        }
        
    }

/*! 
    \param npt_rdata Thermostat/barostat data for rigid bodies 
*/
hipError_t gpu_nph_rigid_reduce_ksum(const gpu_npt_rigid_data& npt_rdata)
    {
    // setup the grid to run the kernel
    int block_size = 128;
    dim3 grid( 1, 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // run the kernel: double the block size to accomodate Ksum_t and Ksum_r
    gpu_nph_rigid_reduce_ksum_kernel<<< grid, threads, 2 * block_size * sizeof(float) >>>(npt_rdata.partial_Ksum_t,
                                                                                          npt_rdata.partial_Ksum_r,
                                                                                          npt_rdata.Ksum_t,
                                                                                          npt_rdata.Ksum_r,
                                                                                          npt_rdata.n_bodies);
    
    return hipSuccess;
    }

