#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: jglaser

#include "TwoStepNPTMTKGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TwoStepNPTMTKGPU.cu
    \brief Defines GPU kernel code for NPT integration on the GPU using the Martyna-Tobias-Klein update equations. Used by TwoStepNPTMTKGPU.
*/

//! Shared memory used in reducing the sum of the squared velocities
extern __shared__ Scalar npt_mtk_sdata[];

//! Kernel to propagate the positions and velocities, first half of NPT update
__global__ void gpu_npt_mtk_step_one_kernel(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar exp_thermo_fac,
                             Scalar mat_exp_v_xx,
                             Scalar mat_exp_v_xy,
                             Scalar mat_exp_v_xz,
                             Scalar mat_exp_v_yy,
                             Scalar mat_exp_v_yz,
                             Scalar mat_exp_v_zz,
                             Scalar mat_exp_v_int_xx,
                             Scalar mat_exp_v_int_xy,
                             Scalar mat_exp_v_int_xz,
                             Scalar mat_exp_v_int_yy,
                             Scalar mat_exp_v_int_yz,
                             Scalar mat_exp_v_int_zz,
                             Scalar mat_exp_r_xx,
                             Scalar mat_exp_r_xy,
                             Scalar mat_exp_r_xz,
                             Scalar mat_exp_r_yy,
                             Scalar mat_exp_r_yz,
                             Scalar mat_exp_r_zz,
                             Scalar mat_exp_r_int_xx,
                             Scalar mat_exp_r_int_xy,
                             Scalar mat_exp_r_int_xz,
                             Scalar mat_exp_r_int_yy,
                             Scalar mat_exp_r_int_yz,
                             Scalar mat_exp_r_int_zz,
                             Scalar deltaT)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // initialize eigenvectors
    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // fetch particle position
        Scalar4 pos = d_pos[idx];

        // fetch particle velocity and acceleration
        Scalar4 vel = d_vel[idx];
        Scalar3 v = make_scalar3(vel.x, vel.y, vel.z);
        Scalar3 accel = d_accel[idx];;
        Scalar3 r = make_scalar3(pos.x, pos.y, pos.z);

        // apply thermostat update of velocity
        v *= exp_thermo_fac;

        // propagate velocity by half a time step and position by the full time step
        // by multiplying with upper triangular matrix
        v.x = mat_exp_v_xx * v.x + mat_exp_v_xy * v.y + mat_exp_v_xz * v.z;
        v.y = mat_exp_v_yy * v.y + mat_exp_v_yz * v.z;
        v.z = mat_exp_v_zz * v.z;

        v.x += mat_exp_v_int_xx * accel.x + mat_exp_v_int_xy * accel.y + mat_exp_v_int_xz * accel.z;
        v.y += mat_exp_v_int_yy * accel.y + mat_exp_v_int_yz * accel.z;
        v.z += mat_exp_v_int_zz * accel.z;

        r.x = mat_exp_r_xx * r.x + mat_exp_r_xy * r.y + mat_exp_r_xz * r.z;
        r.y = mat_exp_r_yy * r.y + mat_exp_r_yz * r.z;
        r.z = mat_exp_r_zz * r.z;

        r.x += mat_exp_r_int_xx * v.x + mat_exp_r_int_xy * v.y + mat_exp_r_int_xz * v.z;
        r.y += mat_exp_r_int_yy * v.y + mat_exp_r_int_yz * v.z;
        r.z += mat_exp_r_int_zz * v.z;

        // write out the results
        d_pos[idx] = make_scalar4(r.x,r.y,r.z,pos.w);
        d_vel[idx] = make_scalar4(v.x,v.y,v.z,vel.w);
        }
    }

/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param exp_thermo_fac Update factor for thermostat
    \param mat_exp_v Matrix exponential for velocity update
    \param mat_exp_v_int Integrated matrix exp for velocity update
    \param mat_exp_r Matrix exponential for position update
    \param mat_exp_r_int Integrated matrix exp for position update
    \param deltaT Time to advance (for one full step)
    \param deltaT Time to move forward in one whole step

    This is just a kernel driver for gpu_npt_mtk_step_one_kernel(). See it for more details.
*/
hipError_t gpu_npt_mtk_step_one(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar exp_thermo_fac,
                             Scalar *mat_exp_v,
                             Scalar *mat_exp_v_int,
                             Scalar *mat_exp_r,
                             Scalar *mat_exp_r_int,
                             Scalar deltaT)
    {
    // setup the grid to run the kernel
    unsigned int block_size = 256;
    dim3 grid( (group_size / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_npt_mtk_step_one_kernel<<< grid, threads >>>(d_pos,
                                                 d_vel,
                                                 d_accel,
                                                 d_group_members,
                                                 group_size,
                                                 exp_thermo_fac,
                                                 mat_exp_v[0],
                                                 mat_exp_v[1],
                                                 mat_exp_v[2],
                                                 mat_exp_v[3],
                                                 mat_exp_v[4],
                                                 mat_exp_v[5],
                                                 mat_exp_v_int[0],
                                                 mat_exp_v_int[1],
                                                 mat_exp_v_int[2],
                                                 mat_exp_v_int[3],
                                                 mat_exp_v_int[4],
                                                 mat_exp_v_int[5],
                                                 mat_exp_r[0],
                                                 mat_exp_r[1],
                                                 mat_exp_r[2],
                                                 mat_exp_r[3],
                                                 mat_exp_r[4],
                                                 mat_exp_r[5],
                                                 mat_exp_r_int[0],
                                                 mat_exp_r_int[1],
                                                 mat_exp_r_int[2],
                                                 mat_exp_r_int[3],
                                                 mat_exp_r_int[4],
                                                 mat_exp_r_int[5],
                                                 deltaT);

    return hipSuccess;
    }

/*! \param N number of particles in the system
    \param d_pos array of particle positions
    \param d_image array of particle images
    \param box The new box the particles where the particles now reside

    Wrap particle positions for all particles in the box
*/
extern "C" __global__
void gpu_npt_mtk_wrap_kernel(const unsigned int N,
                             Scalar4 *d_pos,
                             int3 *d_image,
                             BoxDim box)
    {
    // determine which particle this thread works on
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // wrap ALL particles in the box
    if (idx < N)
        {
        // fetch particle position
        Scalar4 postype = d_pos[idx];
        Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);

        // read in the image flags
        int3 image = d_image[idx];

        // fix periodic boundary conditions
        box.wrap(pos, image);

        // write out the results
        d_pos[idx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
        d_image[idx] = image;
        }
    }

/*! \param N number of particles in the system
    \param d_pos array of particle positions
    \param d_image array of particle images
    \param box The new box the particles where the particles now reside

    This is just a kernel driver for gpu_npt_mtk_wrap_kernel(). See it for more details.
*/
hipError_t gpu_npt_mtk_wrap(const unsigned int N,
                             Scalar4 *d_pos,
                             int3 *d_image,
                             const BoxDim& box)
    {
    // setup the grid to run the kernel
    unsigned int block_size=256;
    dim3 grid( (N / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_npt_mtk_wrap_kernel<<< grid, threads >>>(N, d_pos, d_image, box);

    return hipSuccess;
    }

//! Kernel to propagate the positions and velocities, second half of NPT update
__global__ void gpu_npt_mtk_step_two_kernel(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             const Scalar4 *d_net_force,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar mat_exp_v_xx,
                             Scalar mat_exp_v_xy,
                             Scalar mat_exp_v_xz,
                             Scalar mat_exp_v_yy,
                             Scalar mat_exp_v_yz,
                             Scalar mat_exp_v_zz,
                             Scalar mat_exp_v_int_xx,
                             Scalar mat_exp_v_int_xy,
                             Scalar mat_exp_v_int_xz,
                             Scalar mat_exp_v_int_yy,
                             Scalar mat_exp_v_int_yz,
                             Scalar mat_exp_v_int_zz,
                             Scalar deltaT)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // fetch particle velocity and acceleration
        Scalar4 vel = d_vel[idx];

        // compute acceleration
        Scalar minv = Scalar(1.0)/vel.w;
        Scalar4 net_force = d_net_force[idx];
        Scalar3 accel = make_scalar3(net_force.x, net_force.y, net_force.z);
        accel *= minv;

        Scalar3 v = make_scalar3(vel.x, vel.y, vel.z);

        // propagate velocity by half a time step by multiplying with an upper triangular matrix
        v.x = mat_exp_v_xx * v.x + mat_exp_v_xy * v.y + mat_exp_v_xz * v.z;
        v.y = mat_exp_v_yy * v.y + mat_exp_v_yz * v.z;
        v.z = mat_exp_v_zz * v.z;

        v.x += mat_exp_v_int_xx * accel.x + mat_exp_v_int_xy * accel.y + mat_exp_v_int_xz * accel.z;
        v.y += mat_exp_v_int_yy * accel.y + mat_exp_v_int_yz * accel.z;
        v.z += mat_exp_v_int_zz * accel.z;

        // write out velocity
        d_vel[idx] = make_scalar4(v.x, v.y, v.z, vel.w);

        // since we calculate the acceleration, we need to write it for the next step
        d_accel[idx] = accel;
        }
    }

/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param mat_exp_v Matrix exponential for velocity update
    \param mat_exp_v_int Integrated matrix exp for velocity update
    \param d_net_force Net force on each particle

    \param deltaT Time to move forward in one whole step

    This is just a kernel driver for gpu_npt_mtk_step_kernel(). See it for more details.
*/
hipError_t gpu_npt_mtk_step_two(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar4 *d_net_force,
                             Scalar* mat_exp_v,
                             Scalar* mat_exp_v_int,
                             Scalar deltaT)
    {
    // setup the grid to run the kernel
    unsigned int block_size=256;
    dim3 grid( (group_size / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    gpu_npt_mtk_step_two_kernel<<< grid, threads >>>(d_vel,
                                                     d_accel,
                                                     d_net_force,
                                                     d_group_members,
                                                     group_size,
                                                     mat_exp_v[0],
                                                     mat_exp_v[1],
                                                     mat_exp_v[2],
                                                     mat_exp_v[3],
                                                     mat_exp_v[4],
                                                     mat_exp_v[5],
                                                     mat_exp_v_int[0],
                                                     mat_exp_v_int[1],
                                                     mat_exp_v_int[2],
                                                     mat_exp_v_int[3],
                                                     mat_exp_v_int[4],
                                                     mat_exp_v_int[5],
                                                     deltaT);
        
    return hipSuccess;
    }

//! GPU kernel to perform partial reduction of temperature
__global__ void gpu_npt_mtk_temperature_partial(unsigned int *d_group_members,
                                                unsigned int group_size,
                                                Scalar *d_scratch,
                                                Scalar4 *d_velocity)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    Scalar mv2_element; // element of scratch space read in
    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        Scalar4 vel = d_velocity[idx];
        Scalar mass = vel.w;

        mv2_element =  mass * (vel.x*vel.x + vel.y*vel.y + vel.z*vel.z);
        }
    else
        {
        // non-participating thread: contribute 0 to the sum
        mv2_element = Scalar(0.0);
        }

    npt_mtk_sdata[threadIdx.x] = mv2_element;
    __syncthreads();

    int offs = blockDim.x >> 1;
    while (offs > 0)
        {
        if (threadIdx.x < offs)
            npt_mtk_sdata[threadIdx.x] += npt_mtk_sdata[threadIdx.x + offs];

        offs >>= 1;
        __syncthreads();
        }

    // write out partial sum
    if (threadIdx.x == 0)
        d_scratch[blockIdx.x] = npt_mtk_sdata[0];

     }

//! GPU kernel to perform final reduction of temperature
__global__ void gpu_npt_mtk_temperature_final_sum(Scalar *d_scratch,
                                                  Scalar *d_temperature,
                                                  unsigned int ndof,
                                                  unsigned int num_partial_sums)
    {
    Scalar final_sum(0.0);

    for (int start = 0; start < num_partial_sums; start += blockDim.x)
        {
        __syncthreads();
        if (start + threadIdx.x < num_partial_sums)
            {
            npt_mtk_sdata[threadIdx.x] = d_scratch[start + threadIdx.x];
            }
        else
            npt_mtk_sdata[threadIdx.x] = Scalar(0.0);

        __syncthreads();

        // reduce the sum in parallel
        int offs = blockDim.x >> 1;
        while (offs > 0)
            {
            if (threadIdx.x < offs)
                npt_mtk_sdata[threadIdx.x] += npt_mtk_sdata[threadIdx.x + offs];

            offs >>=1;
            __syncthreads();
            }

        if (threadIdx.x == 0)
            final_sum += npt_mtk_sdata[0];
        }

    if (threadIdx.x == 0)
        *d_temperature = final_sum/Scalar(ndof);
    }

/*!\param d_temperature Device variable to store the temperature value (output)
   \param d_vel Array of particle velocities and masses
   \param d_scratch Temporary scratch space for reduction
   \param num_blocks Number of CUDA blocks used in reduction
   \param block_size Size of blocks used in reduction
   \param d_group_members Members of group for which the reduction is performed
   \param group_size Size of group
   \param ndof Number of degrees of freedom of group

   This function performs the reduction of the temperature on the GPU. It is just
   a driver function that calls the appropriate GPU kernels.
   */
hipError_t gpu_npt_mtk_temperature(Scalar *d_temperature,
                                    Scalar4 *d_vel,
                                    Scalar *d_scratch,
                                    unsigned int num_blocks,
                                    unsigned int block_size,
                                    unsigned int *d_group_members,
                                    unsigned int group_size,
                                    unsigned int ndof)
    {
    assert(d_temperature);
    assert(d_vel);
    assert(d_group_members);
    assert(d_scratch);

    dim3 grid(num_blocks,1,1);
    dim3 threads(block_size,1,1);

    unsigned int shared_bytes = sizeof(Scalar)*block_size;

    // reduce squared velocity norm times mass, first pass
    gpu_npt_mtk_temperature_partial<<<grid, threads, shared_bytes>>>(
                                                d_group_members,
                                                group_size,
                                                d_scratch,
                                                d_vel);


    unsigned int final_block_size = 512;
    grid = dim3(1,1,1);
    threads = dim3(final_block_size, 1, 1);
    shared_bytes = sizeof(Scalar)*final_block_size;

    // reduction, second pass
    gpu_npt_mtk_temperature_final_sum<<<grid, threads, shared_bytes>>>(
                                                d_scratch,
                                                d_temperature,
                                                ndof,
                                                num_blocks);

    return hipSuccess;
    }

/*! \param d_vel array of particle velocities and masses
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param exp_v_fac_thermo scaling factor (per direction) for velocity update generated by thermostat

    GPU kernel to thermostat velocities
*/
__global__ void gpu_npt_mtk_thermostat_kernel(Scalar4 *d_vel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar exp_v_fac_thermo)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // fetch particle velocity and acceleration
        Scalar4 vel = d_vel[idx];
        Scalar3 v = make_scalar3(vel.x, vel.y, vel.z);

        v = v*exp_v_fac_thermo;

        // write out the results
        d_vel[idx] = make_scalar4(v.x,v.y,v.z,vel.w);

        }
    }

/*! \param d_vel array of particle velocities
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param xi Thermostat velocity
    \param deltaT Time to move forward in one whole step

    This is just a kernel driver for gpu_npt_step_kernel(). See it for more details.
*/
hipError_t gpu_npt_mtk_thermostat(Scalar4 *d_vel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar xi,
                             Scalar deltaT)
    {
    // setup the grid to run the kernel
    unsigned int block_size=256;
    dim3 grid( (group_size / block_size) + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    Scalar exp_v_fac_thermo = exp(-Scalar(1.0/2.0)*xi*deltaT);

    // run the kernel
    gpu_npt_mtk_thermostat_kernel<<< grid, threads >>>(d_vel,
                                                     d_group_members,
                                                     group_size,
                                                     exp_v_fac_thermo);

    return hipSuccess;
    }
