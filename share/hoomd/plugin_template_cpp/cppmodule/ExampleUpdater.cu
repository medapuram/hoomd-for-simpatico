#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "ExampleUpdater.cuh"

/*! \file ExampleUpdater.cu
    \brief CUDA kernels for ExampleUpdater
*/

// First, the kernel code for zeroing the velocities on the GPU
//! Kernel that zeroes velocities on the GPU
/*! \param d_vel Velocity-mass array from the ParticleData
    \param N Number of particles
    
    This kernel executes one thread per particle and zeros the velocity of each. It can be run with any 1D block size
    as long as block_size * num_blocks is >= the number of particles.
*/
extern "C" __global__ 
void gpu_zero_velocities_kernel(float4 *d_vel, unsigned int N)
    {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
        {
        // vel.w is the mass, don't want to modify that
        float4 vel = d_vel[idx];
        vel.x = vel.y = vel.z = 0.0f;
        d_vel[idx] = vel;
        }
    }

/*! \param d_vel Velocity-mass array from the ParticleData
    \param N Number of particles
    This is just a driver for gpu_zero_velocities_kernel(), see it for the details
*/
hipError_t gpu_zero_velocities(float4 *d_vel, unsigned int N)
    {
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    
    // run the kernel
    gpu_zero_velocities_kernel<<< grid, threads >>>(d_vel, N);
    
    // this method always succeds. If you had a cuda* call in this driver, you could return its error code if not
    // hipSuccess
    return hipSuccess;
    }

